#include "hip/hip_runtime.h"
#include "common.h"
#include "hipsparse.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <algorithm>

#include "common.h"

using namespace std;

#define OFFSET(row, col, ld) ((row) * ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid + 8]; 
    sdata[tid] += sdata[tid + 4]; 
    sdata[tid] += sdata[tid + 2]; 
    sdata[tid] += sdata[tid + 1]; 
}

__device__ __forceinline__ const int* add_ptr_u(const int* src, int offset)      \
{                                                                            \
    const int* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ const float* add_ptr_f(const float* src, int offset)      \
{                                                                            \
    const float* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}



__global__ void convert_bcsr_kernel_transpose_1(const int * __restrict__  mask, int* __restrict__ row_ptr, int* __restrict__ col_idx, int * extra_buffer, int n_block_h, int n_block_w)
{

    
    // initial the shared flag
    uint bx = blockIdx.x;
    uint tid = threadIdx.x;
    uint step = blockDim.x;

    int pos_id;
    int flag;
    int global_offset;
    for(int by=tid; by<n_block_h;by+=step){
        global_offset =  (by * n_block_w) + bx;
        flag = mask[global_offset];
        if(flag>0){
            pos_id= atomicAdd(&extra_buffer[bx], 1);
            atomicAdd(&extra_buffer[bx+n_block_w], 1);
            atomicAdd(&row_ptr[n_block_w], 1);
            extra_buffer[2*n_block_w + n_block_h * bx + pos_id] = by;
        }
    
    }


}
__global__ void convert_bcsr_kernel_transpose_2(const int * __restrict__  mask, int* __restrict__ row_ptr, int* __restrict__ col_idx, int * extra_buffer, int n_block_h, int n_block_w)
{
    uint bx = blockIdx.x;
    uint tid = threadIdx.x;
    int pos_id, ori_bx, ori_by;
    __shared__ int prefix_count;
    __shared__ int remain_count;
    if(tid==0){
        prefix_count = 0;
        for(int i=0; i<bx;i++){
            prefix_count +=  extra_buffer[n_block_w+i];
        }
        remain_count = extra_buffer[n_block_w+bx];
        row_ptr[bx] = prefix_count;
    }
    __syncthreads();
    for(int tmp=tid; tmp<remain_count;tmp+=blockDim.x){
        pos_id = atomicSub(&extra_buffer[bx], 1);
        pos_id-=1;
        if(pos_id>=0){
            ori_bx = bx;
            ori_by = extra_buffer[ 2*n_block_w + bx * n_block_h + pos_id];       
            row_ptr[bx] = prefix_count;
            col_idx[prefix_count + pos_id] = ori_by;
        }
    }


}


__global__ void convert_bcsr_kernel_1(const int * __restrict__  mask, int* __restrict__ row_ptr, int* __restrict__ col_idx, int * extra_buffer, int n_block_h, int n_block_w)
{

    
    // initial the shared flag
    uint by = blockIdx.x;
    uint tid = threadIdx.x;
    uint step = blockDim.x;

    int pos_id;
    int flag;
    int global_offset;
    for(int bx=tid; bx<n_block_w;bx+=step){
        global_offset =  (by * n_block_w) + bx;
        flag = mask[global_offset];
        if(flag>0){
            pos_id= atomicAdd(&extra_buffer[by], 1);
            atomicAdd(&extra_buffer[by+n_block_h], 1);
            atomicAdd(&row_ptr[n_block_h], 1);
            extra_buffer[2*n_block_h + n_block_w * by + pos_id] = bx;
        }
    
    }


}
__global__ void convert_bcsr_kernel_2(const int * __restrict__  mask, int* __restrict__ row_ptr, int* __restrict__ col_idx, int * extra_buffer, int n_block_h, int n_block_w)
{
    uint by = blockIdx.x;
    uint tid = threadIdx.x;
    int pos_id, ori_bx, ori_by;
    __shared__ int prefix_count;
    __shared__ int remain_count;
    if(tid==0){
        prefix_count = 0;
        for(int i=0; i<by;i++){
            prefix_count +=  extra_buffer[n_block_h+i];
        }
        remain_count = extra_buffer[n_block_h+by];
        row_ptr[by] = prefix_count;
    }
    __syncthreads();
    for(int tmp=tid; tmp<remain_count;tmp+=blockDim.x){
        pos_id = atomicSub(&extra_buffer[by], 1);
        pos_id-=1;
        if(pos_id>=0){
            ori_by = by;
            ori_bx = extra_buffer[ 2*n_block_h + by * n_block_w + pos_id];       
            row_ptr[by] = prefix_count;
            col_idx[prefix_count + pos_id] = ori_bx;
        }
    }


}
void convert_bcsr_transpose(int * mask, int * row_ptr, int * col_idx, int * ext_buffer, 
                            int n_block_h, int n_block_w)
{
    // the mask is a binary matrix with shape of n_block_h x n_block_w
    // build the csr index along the n_block_w
    // need reset the extra buffer here
    CUDA_SAFE_CALL(hipMemset((void*)ext_buffer, 0, sizeof(int)*(2*n_block_w+n_block_h*n_block_w)) );
    CUDA_SAFE_CALL(hipMemset((void*)row_ptr, 0, sizeof(int)*(1+(n_block_w))) );
    dim3 block_dim(128);
    dim3 grid_dim(n_block_w);

    convert_bcsr_kernel_transpose_1<<<grid_dim, block_dim>>>(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
    convert_bcsr_kernel_transpose_2<<<grid_dim, block_dim>>>(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
}


void convert_bcsr(int * mask, int * row_ptr, int * col_idx, int * ext_buffer, 
                            int n_block_h, int n_block_w)
{

    CUDA_SAFE_CALL(hipMemset((void*)ext_buffer, 0, sizeof(int)*(2*n_block_h+n_block_h*n_block_w)) );
    CUDA_SAFE_CALL(hipMemset((void*)row_ptr, 0, sizeof(int)*(1+(n_block_h))) );
    dim3 block_dim(128);
    dim3 grid_dim(n_block_h);

    convert_bcsr_kernel_1<<<grid_dim, block_dim>>>(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
    convert_bcsr_kernel_2<<<grid_dim, block_dim>>>(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
}

void convert_bcsr_blockwise(int * mask, int *row_ptr, int * col_idx, int* ext_buffer, int n_block_h, int n_block_w, int transpose)
{
    if(transpose){
        convert_bcsr_transpose(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
    } else{
        // TO BE DONE
        convert_bcsr(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
    }

}
std::vector<at::Tensor> convert_bcsr_forward_blockwise(
    torch::Tensor sparse_pattern,
    int transpose)
{
    int n_block_h = sparse_pattern.size(0);
    int n_block_w = sparse_pattern.size(1);
    int row_size = transpose? n_block_w: n_block_h;
    int n_total_block = n_block_h * n_block_w;
    torch::Tensor csr_row = torch::zeros({row_size+1}, sparse_pattern.options());
    torch::Tensor csr_col = torch::zeros({n_total_block}, sparse_pattern.options());
    torch::Tensor ext_buffer = torch::zeros({2*row_size+n_total_block}, sparse_pattern.options());
    AT_DISPATCH_INTEGRAL_TYPES(sparse_pattern.type(), "convert_bcsr_blockwise", ([&]
        { convert_bcsr_blockwise(
                sparse_pattern.data_ptr<int>(),
                csr_row.data_ptr<int>(),
                csr_col.data_ptr<int>(),
                ext_buffer.data_ptr<int>(),
                n_block_h,
                n_block_w,
                transpose
            ); }));
    std::vector<torch::Tensor> bcsr({csr_row, csr_col});
    return bcsr;
}
