#include "hip/hip_runtime.h"
#include "common.h"
#include "hipsparse.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <algorithm>

#include "common.h"

using namespace std;

#define OFFSET(row, col, ld) ((row) * ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid + 8]; 
    sdata[tid] += sdata[tid + 4]; 
    sdata[tid] += sdata[tid + 2]; 
    sdata[tid] += sdata[tid + 1]; 
}

__device__ __forceinline__ const int* add_ptr_u(const int* src, int offset)      \
{                                                                            \
    const int* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ const float* add_ptr_f(const float* src, int offset)      \
{                                                                            \
    const float* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__global__ void convert_bcsr_kernel_1(const int * __restrict__  mask, float * __restrict__  dense, int h, int w,
                                int block_h, int block_w, int * row, int *col, int * row_pos,
                                float * values, int * extra_buffer)
{

    __shared__ int reduce[MAX_BLOCK_THREAD_COUNT];
    assert(blockDim.x<=MAX_BLOCK_THREAD_COUNT);
    // initial the shared flag
    uint bx = blockIdx.x;
    uint by = blockIdx.y;
    uint tid = threadIdx.x;
    int global_offset =  (by * block_h) * w + bx * block_w;
    int block_size =  block_h * block_w;
    assert(block_w % 4 == 0);
    // cannot handle the misalignment for now
    assert((block_size / 4) % blockDim.x==0);
    int flag = 0;
    for(int _pos = tid; _pos< block_size / 4; _pos+=blockDim.x){
        uint block_offset = _pos / (block_w / 4) * w + _pos % (block_w / 4) * 4;        
        int4 data = __ldg((const int4*)(add_ptr_u(mask, global_offset+block_offset)));
        flag += data.x + data.y + data.z + data.w;
    }
    reduce[tid] = flag;
    __syncthreads();
    // fast tree reduce accross the block
    for(uint s=blockDim.x/2; s>32; s>>=1){
        if(tid<s)
            reduce[tid] += reduce[tid+s];
        __syncthreads();
    }
    if(tid<32)
        warpReduce(reduce, tid);
    __syncthreads();
    int pos_id;
    if(tid==0 && reduce[0]>0){
        pos_id= atomicAdd(&extra_buffer[by], 1);
        atomicAdd(&extra_buffer[by+h], 1);
        atomicAdd(&row[h/block_h], 1);
        extra_buffer[2*h + gridDim.x * by + pos_id] = bx;
    }

}
__global__ void convert_bcsr_kernel_2(const int * __restrict__  mask, float * __restrict__  dense, int h, int w,
    int block_h, int block_w, int * row, int *col, int * row_pos,
    float * values, int * extra_buffer, int * block_index)
{
    __shared__ int pos_id, prefix_count, ori_bx, ori_by;
    __shared__ int prefix_sum[MAX_BLOCK_THREAD_COUNT];
    uint by = blockIdx.y;
    uint bx = blockIdx.x;
    uint tid = threadIdx.x;

    if (tid==0){
        pos_id = -1;
        prefix_count = 0;
        // contend for the block

        pos_id = atomicSub(&extra_buffer[by], 1);
        pos_id-=1;
        if (pos_id>=0){
            for(int i=0; i<by;i++){
                prefix_count +=  extra_buffer[h+i];
            }
            ori_by = by;
            ori_bx = extra_buffer[ 2*h + by * gridDim.x + pos_id];       
            
            row[by] = prefix_count;
            col[prefix_count+pos_id] = ori_bx;
            row_pos[prefix_count+pos_id] = by;
            block_index[by*gridDim.x+ori_bx] = prefix_count+pos_id;
        }
        else if(pos_id==-1){
            for(int i=0; i<by;i++){
                prefix_count +=  extra_buffer[h+i];
            }            
            row[by] = prefix_count;
        }
    }
    __syncthreads();
    if(pos_id>=0){
        int global_offset =  (ori_by * block_h) * w + ori_bx * block_w;
        int block_size = block_h * block_w;
        int write_global_offset = (prefix_count + pos_id) * block_size;

        for(int _pos=tid; _pos<block_size/4; _pos+=blockDim.x){
            uint block_offset = _pos / (block_w / 4) * w + _pos % (block_w / 4) * 4;
            float4 data = __ldg((const float4*)(add_ptr_f(dense, global_offset + block_offset)));
            *(float4*)&values[write_global_offset+_pos*4] = data;
        }
        
    }

}

void convert_bcsr()
{
    
}


__global__ void convert_bcsr_kernel_transpose_1(const int * __restrict__  mask, int* __restrict__ row_ptr, int* __restrict__ col_idx, int * extra_buffer, int n_block_h, int n_block_w)
{

    
    // initial the shared flag
    uint bx = blockIdx.x;
    uint by = blockIdx.y;
    uint tid = threadIdx.x;

    int global_offset =  (by * n_block_w) + bx;
    int pos_id;
    int flag = mask[global_offset];
    if(tid==0 && flag>0){
        pos_id= atomicAdd(&extra_buffer[bx], 1);
        atomicAdd(&extra_buffer[bx+n_block_w], 1);
        atomicAdd(&row_ptr[n_block_w], 1);
        // printf("block nnz: %d\n", tmp);
        // extra_buffer[2*w + gridDim.x * by + pos_id] = bx;
        extra_buffer[2*n_block_w + gridDim.y * bx + pos_id] = by;
    }

}
__global__ void convert_bcsr_kernel_transpose_2(const int * __restrict__  mask, int* __restrict__ row_ptr, int* __restrict__ col_idx, int * extra_buffer, int n_block_h, int n_block_w)
{
    uint by = blockIdx.y;
    uint bx = blockIdx.x;
    uint tid = threadIdx.x;
    int pos_id, prefix_count, ori_bx, ori_by;
    __shared__ int prefix_sum[MAX_BLOCK_THREAD_COUNT];
    if (tid==0){
        pos_id = -1;
        prefix_count = 0;
        // contend for the block

        pos_id = atomicSub(&extra_buffer[bx], 1);
        pos_id-=1;
        if (pos_id>=0){
            for(int i=0; i<bx;i++){
                prefix_count +=  extra_buffer[n_block_w+i];
            }
            ori_bx = bx;
            ori_by = extra_buffer[ 2*n_block_w + bx * gridDim.y + pos_id];       
            
            row_ptr[bx] = prefix_count;
            col_idx[prefix_count + pos_id] = ori_by;
        }
        else if(pos_id==-1){
            for(int i=0; i<bx; i++){
                prefix_count +=  extra_buffer[n_block_w+i];
            }            
            row_ptr[bx] = prefix_count;
        }
    }


}
void convert_bcsr_transpose(int * mask, int * row_ptr, int * col_idx, int * ext_buffer, 
                            int n_block_h, int n_block_w)
{
    // the mask is a binary matrix with shape of n_block_h x n_block_w
    // build the csr index along the n_block_w
    // need reset the extra buffer here
    CUDA_SAFE_CALL(hipMemset((void*)ext_buffer, 0, sizeof(int)*(2*n_block_w+n_block_h*n_block_w)) );
    CUDA_SAFE_CALL(hipMemset((void*)row_ptr, 0, sizeof(int)*(1+(n_block_w))) );
    dim3 block_dim(1);
    dim3 grid_dim(n_block_w, n_block_h);

    convert_bcsr_kernel_transpose_1<<<grid_dim, block_dim>>>(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
    convert_bcsr_kernel_transpose_2<<<grid_dim, block_dim>>>(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
}

void convert_bcsr_blockwise(int * mask, int *row_ptr, int * col_idx, int* ext_buffer, int n_block_h, int n_block_w, int transpose)
{
    if(transpose){
        convert_bcsr_transpose(mask, row_ptr, col_idx, ext_buffer, n_block_h, n_block_w);
    } else{
        // TO BE DONE
        convert_bcsr();
    }

}
std::vector<at::Tensor> convert_bcsr_forward_blockwise(
    torch::Tensor sparse_pattern,
    int transpose)
{
    int n_block_h = sparse_pattern.size(0);
    int n_block_w = sparse_pattern.size(1);
    int row_size = transpose? n_block_w: n_block_h;
    int n_total_block = n_block_h * n_block_w;
    torch::Tensor csr_row = torch::zeros({row_size+1}, sparse_pattern.options());
    torch::Tensor csr_col = torch::zeros({n_total_block}, sparse_pattern.options());
    torch::Tensor ext_buffer = torch::zeros({2*row_size+n_total_block}, sparse_pattern.options());
    AT_DISPATCH_INTEGRAL_TYPES(sparse_pattern.type(), "convert_bcsr_blockwise", ([&]
        { convert_bcsr_blockwise(
                sparse_pattern.data_ptr<int>(),
                csr_row.data_ptr<int>(),
                csr_col.data_ptr<int>(),
                ext_buffer.data_ptr<int>(),
                n_block_h,
                n_block_w,
                transpose
            ); }));
    std::vector<torch::Tensor> bcsr({csr_row, csr_col});
    return bcsr;
}
