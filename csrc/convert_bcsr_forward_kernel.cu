#include "hip/hip_runtime.h"
#include "common.h"
#include "hipsparse.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <algorithm>

#include "common.h"

using namespace std;

#define OFFSET(row, col, ld) ((row) * ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid + 8]; 
    sdata[tid] += sdata[tid + 4]; 
    sdata[tid] += sdata[tid + 2]; 
    sdata[tid] += sdata[tid + 1]; 
}

__device__ __forceinline__ const int* add_ptr_u(const int* src, int offset)      \
{                                                                            \
    const int* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ const float* add_ptr_f(const float* src, int offset)      \
{                                                                            \
    const float* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__global__ void convert_bcsr_kernel_1(const int * __restrict__  mask, float * __restrict__  dense, int h, int w,
                                int block_h, int block_w, int * row, int *col, int * row_pos,
                                float * values, int * extra_buffer)
{

    __shared__ int reduce[MAX_BLOCK_THREAD_COUNT];
    assert(blockDim.x<=MAX_BLOCK_THREAD_COUNT);
    // initial the shared flag
    uint bx = blockIdx.x;
    uint by = blockIdx.y;
    uint tid = threadIdx.x;
    int global_offset =  (by * block_h) * w + bx * block_w;
    int block_size =  block_h * block_w;
    assert(block_w % 4 == 0);
    // cannot handle the misalignment for now
    assert((block_size / 4) % blockDim.x==0);
    int flag = 0;
    for(int _pos = tid; _pos< block_size / 4; _pos+=blockDim.x){
        uint block_offset = _pos / (block_w / 4) * w + _pos % (block_w / 4) * 4;        
        int4 data = __ldg((const int4*)(add_ptr_u(mask, global_offset+block_offset)));
        flag += data.x + data.y + data.z + data.w;
    }
    reduce[tid] = flag;
    __syncthreads();
    // fast tree reduce accross the block
    for(uint s=blockDim.x/2; s>32; s>>=1){
        if(tid<s)
            reduce[tid] += reduce[tid+s];
        __syncthreads();
    }
    if(tid<32)
        warpReduce(reduce, tid);
    __syncthreads();
    int pos_id;
    if(tid==0 && reduce[0]>0){
        pos_id= atomicAdd(&extra_buffer[by], 1);
        atomicAdd(&extra_buffer[by+h], 1);
        atomicAdd(&row[h/block_h], 1);
        extra_buffer[2*h + gridDim.x * by + pos_id] = bx;
    }

}
__global__ void convert_bcsr_kernel_2(const int * __restrict__  mask, float * __restrict__  dense, int h, int w,
    int block_h, int block_w, int * row, int *col, int * row_pos,
    float * values, int * extra_buffer)
{
    __shared__ int pos_id, prefix_count, ori_bx, ori_by;
    __shared__ int prefix_sum[MAX_BLOCK_THREAD_COUNT];
    uint by = blockIdx.y;
    uint bx = blockIdx.x;
    uint tid = threadIdx.x;

    if (tid==0){
        pos_id = -1;
        prefix_count = 0;
        // contend for the block

        pos_id = atomicSub(&extra_buffer[by], 1);
        pos_id-=1;
        if (pos_id>=0){
            for(int i=0; i<by;i++){
                prefix_count +=  extra_buffer[h+i];
            }
            ori_by = by;
            ori_bx = extra_buffer[ 2*h + by * gridDim.x + pos_id];       
            
            row[by] = prefix_count;
            col[prefix_count+pos_id] = ori_bx;
            row_pos[prefix_count+pos_id] = by;
        }
    }
    __syncthreads();
    if(pos_id>=0){
        int global_offset =  (ori_by * block_h) * w + ori_bx * block_w;
        int block_size = block_h * block_w;
        int write_global_offset = (prefix_count + pos_id) * block_size;

        for(int _pos=tid; _pos<block_size/4; _pos+=blockDim.x){
            uint block_offset = _pos / (block_w / 4) * w + _pos % (block_w / 4) * 4;
            float4 data = __ldg((const float4*)(add_ptr_f(dense, global_offset + block_offset)));
            *(float4*)&values[write_global_offset+_pos*4] = data;
        }
        
    }

}

void convert_bcsr(int * mask, float * dense, int h, int w,
    int block_h, int block_w, int * row, int *col, int * row_pos,
    float*values, int * extra_buffer)
{
    // need reset the extra buffer here
    assert(block_w % 4 == 0);
    CUDA_SAFE_CALL(hipMemset((void*)extra_buffer, 0, sizeof(int)*(2*h+(h/block_h)*(w/block_w))) );
    CUDA_SAFE_CALL(hipMemset((void*)row, 0, sizeof(int)*(1+(h/block_h))) );
    dim3 block_dim(block_h*block_w/4);
    dim3 grid_dim(w/block_w, h/block_h);
    // std::cout<<"grid_dim "<< w/block_w << ", " <<h/block_h << std::endl;
    convert_bcsr_kernel_1<<<grid_dim, block_dim>>>(mask, dense, h, w, block_h, block_w, row, col, row_pos, values, extra_buffer);
    convert_bcsr_kernel_2<<<grid_dim, block_dim>>>(mask, dense, h, w, block_h, block_w, row, col, row_pos, values, extra_buffer);


}

std::vector<at::Tensor> convert_bcsr_forward(
    torch::Tensor sparse_pattern,
    torch::Tensor dense_values,
    int block_h, 
    int block_w)
{
    int h = sparse_pattern.size(0);
    int w = sparse_pattern.size(1);
    assert(h % block_h==0);
    assert(w % block_w==0);
    // allocate enough memory for the sparse values
    torch::Tensor csr_values = torch::empty_like(dense_values);
    torch::Tensor csr_row = torch::zeros({h/block_h+1}, sparse_pattern.options());
    int n_total_block = h * w / block_h / block_w;
    torch::Tensor csr_row_pos = torch::zeros({n_total_block}, sparse_pattern.options());
    torch::Tensor csr_col = torch::zeros({n_total_block}, sparse_pattern.options());
    torch::Tensor ext_buffer = torch::zeros({2*h+n_total_block}, sparse_pattern.options());
    
    AT_DISPATCH_FLOATING_TYPES(dense_values.type(), "convert_bcsr", ([&]
        { convert_bcsr(
                sparse_pattern.data_ptr<int>(),
                dense_values.data_ptr<float>(),
                h, w, block_h, block_w,
                csr_row.data_ptr<int>(),
                csr_col.data_ptr<int>(),
                csr_row_pos.data_ptr<int>(),
                csr_values.data_ptr<float>(),
                ext_buffer.data_ptr<int>()
            ); }));
    std::vector<torch::Tensor> bcsr({csr_row, csr_col, csr_row_pos, csr_values});
    return bcsr;
}
