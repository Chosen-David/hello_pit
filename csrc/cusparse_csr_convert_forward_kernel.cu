#include "common.h"
#include "hipsparse.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>

using namespace std;
// Macro definition for the cuda and cusparse
// cuSparse SPMM interface
int cusparse_spmm(
    int M,
    int K,
    int N,
    int * row_idx,
    int * col_idx,
    float * values,
    float * MB,
    float * MC,
    float * alpha,
    float * beta
);
#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)
#define CUSPARSE_SAFE_CALL(func)                                                                \
    do                                                                                          \
    {                                                                                           \
        hipsparseStatus_t e = (func);                                                            \
        if (e != HIPSPARSE_STATUS_SUCCESS)                                                       \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

int cusparse_csr_convert(
    float* dense_value,
    int n_row,
    int n_col,
    int * csr_row,
    int * csr_col,
    float * csr_val)
{
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matB;
    hipsparseDnMatDescr_t matA;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CUSPARSE_SAFE_CALL(hipsparseCreate(&handle));

    CUSPARSE_SAFE_CALL(hipsparseCreateDnMat(&matA, n_row, n_col, n_col, dense_value,
                                    HIP_R_32F, HIPSPARSE_ORDER_ROW));
    CUSPARSE_SAFE_CALL( hipsparseCreateCsr(&matB, n_row, n_col, 0,
                                    csr_row, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_bufferSize(
                                        handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) );
    CUDA_SAFE_CALL( hipMalloc(&dBuffer, bufferSize) );
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_analysis(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) );
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CUSPARSE_SAFE_CALL( hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,
                                        &nnz) );
    // torch::Tensor csr_col = torch::empty_like({nnz}, csr_row);
    // torch::Tensor csr_values = torch::empty_like({nnz}, dense_values);
    CUSPARSE_SAFE_CALL( hipsparseCsrSetPointers(matB, csr_row, csr_col, csr_val) );
    // execute Sparse to Dense conversion
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_convert(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) );
    CUSPARSE_SAFE_CALL( hipsparseDestroyDnMat(matA) );
    CUSPARSE_SAFE_CALL( hipsparseDestroySpMat(matB) );
    CUSPARSE_SAFE_CALL( hipsparseDestroy(handle) );
}

std::vector<at::Tensor> cusparse_convert_forward(
    torch::Tensor dense_values)
{   
    hipSetDevice(dense_values.get_device());
    // the weight shape should be KxN
    int n_row = dense_values.size(0);
    int n_col = dense_values.size(1);
    auto csr_row_options =
    torch::TensorOptions().dtype(torch::kInt32).device(dense_values.options().device());
    torch::Tensor csr_row = torch::empty({n_row+1}, csr_row_options);
    torch::Tensor csr_col = torch::empty({n_row*n_col}, csr_row_options);
    torch::Tensor csr_val = torch::empty_like(dense_values);
    AT_DISPATCH_FLOATING_TYPES(dense_values.type(), "cusparse convert_bcsr", ([&]
    { cusparse_csr_convert(
            dense_values.data_ptr<float>(),
            n_row,
            n_col,
            csr_row.data_ptr<int>(),
            csr_col.data_ptr<int>(),
            csr_val.data_ptr<float>()
        ); }));
    std::vector<at::Tensor> csr({csr_row, csr_col, csr_val});
    return csr;
}
