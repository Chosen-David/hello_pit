#include "common.h"
#include "hipsparse.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>

using namespace std;
// Macro definition for the cuda and cusparse
// cuSparse SPMM interface

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)
#define CUSPARSE_SAFE_CALL(func)                                                                \
    do                                                                                          \
    {                                                                                           \
        hipsparseStatus_t e = (func);                                                            \
        if (e != HIPSPARSE_STATUS_SUCCESS)                                                       \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

int cusparse_csr_convert(
    float* dense_value,
    int n_row,
    int n_col,
    int * csr_row,
    int * csr_col,
    float * csr_val)
{
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matB;
    hipsparseDnMatDescr_t matA;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CUSPARSE_SAFE_CALL(hipsparseCreate(&handle));

    CUSPARSE_SAFE_CALL(hipsparseCreateDnMat(&matA, n_row, n_col, n_col, dense_value,
                                    HIP_R_32F, HIPSPARSE_ORDER_ROW));
    CUSPARSE_SAFE_CALL( hipsparseCreateCsr(&matB, n_row, n_col, 0,
                                    csr_row, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_bufferSize(
                                        handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) );
    CUDA_SAFE_CALL( hipMalloc(&dBuffer, bufferSize) );
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_analysis(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) );
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CUSPARSE_SAFE_CALL( hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,
                                        &nnz) );
    // torch::Tensor csr_col = torch::empty_like({nnz}, csr_row);
    // torch::Tensor csr_values = torch::empty_like({nnz}, dense_values);
    CUSPARSE_SAFE_CALL( hipsparseCsrSetPointers(matB, csr_row, csr_col, csr_val) );
    // execute Sparse to Dense conversion
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_convert(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) );
    CUSPARSE_SAFE_CALL( hipsparseDestroyDnMat(matA) );
    CUSPARSE_SAFE_CALL( hipsparseDestroySpMat(matB) );
    CUSPARSE_SAFE_CALL( hipsparseDestroy(handle) );
}

std::vector<at::Tensor> cusparse_convert_forward(
    torch::Tensor dense_values)
{   
    hipSetDevice(dense_values.get_device());
    // the weight shape should be KxN
    int n_row = dense_values.size(0);
    int n_col = dense_values.size(1);
    auto csr_row_options =
    torch::TensorOptions().dtype(torch::kInt32).device(dense_values.options().device());
    torch::Tensor csr_row = torch::empty({n_row+1}, csr_row_options);
    torch::Tensor csr_col = torch::empty({n_row*n_col}, csr_row_options);
    torch::Tensor csr_val = torch::empty_like(dense_values);
    AT_DISPATCH_FLOATING_TYPES(dense_values.type(), "cusparse convert_bcsr", ([&]
    { cusparse_csr_convert(
            dense_values.data_ptr<float>(),
            n_row,
            n_col,
            csr_row.data_ptr<int>(),
            csr_col.data_ptr<int>(),
            csr_val.data_ptr<float>()
        ); }));
    std::vector<at::Tensor> csr({csr_row, csr_col, csr_val});
    return csr;
}

void cusparse_csr_sparse_to_dense(int num_rows,
                                  int num_cols,
                                  int nnz,
                                  int * d_csr_offsets,
                                  int * d_csr_columns,
                                  float * d_csr_values,
                                  float * d_dense
                                    )
{
    // // CUSPARSE APIs
    // int num_rows = dense_shape[0];
    // int num_cols = dense_shape[1];

    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CUSPARSE_SAFE_CALL( hipsparseCreate(&handle) );
    // Create sparse matrix A in CSR format
    CUSPARSE_SAFE_CALL( hipsparseCreateCsr(&matA, num_rows, num_cols, nnz,
                                      d_csr_offsets, d_csr_columns,
                                      d_csr_values, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
    // Create dense matrix B
    CUSPARSE_SAFE_CALL( hipsparseCreateDnMat(&matB, num_rows, num_cols, num_cols, d_dense,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) );
    // allocate an external buffer if needed
    CUSPARSE_SAFE_CALL( hipsparseSparseToDense_bufferSize(
                                        handle, matA, matB,
                                        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                        &bufferSize) );
    CUDA_SAFE_CALL( hipMalloc(&dBuffer, bufferSize) );

    // execute Sparse to Dense conversion
    CUSPARSE_SAFE_CALL( hipsparseSparseToDense(handle, matA, matB,
                                          HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                          dBuffer) );
    // destroy matrix/vector descriptors
    CUDA_SAFE_CALL( hipFree(dBuffer) );
    CUSPARSE_SAFE_CALL( hipsparseDestroySpMat(matA) );
    CUSPARSE_SAFE_CALL( hipsparseDestroyDnMat(matB) );
    CUSPARSE_SAFE_CALL( hipsparseDestroy(handle) );

}

at::Tensor cusparse_convert_backward(
    torch::Tensor csr_row,
    torch::Tensor csr_col,
    torch::Tensor csr_val,
    int n_row,
    int n_col,
    int nnz
){
    hipSetDevice(csr_row.get_device());
    // the weight shape should be KxN
    // assert( n_row == csr_row.size(0));
    torch::Tensor dense_out = torch::empty({n_row, n_col}, csr_val.options());
    AT_DISPATCH_FLOATING_TYPES(csr_val.type(), "cusparse convert_csr", ([&]
    { cusparse_csr_sparse_to_dense(
            n_row,
            n_col,
            nnz,
            csr_row.data_ptr<int>(),
            csr_col.data_ptr<int>(),
            csr_val.data_ptr<float>(),
            dense_out.data_ptr<float>()
        ); }));
    
    return dense_out;

}
