#include "common.h"
#include "hipsparse.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>

using namespace std;
// Macro definition for the cuda and cusparse
// cuSparse SPMM interface
int cusparse_spmm(
    int M,
    int K,
    int N,
    int * row_idx,
    int * col_idx,
    float * values,
    float * MB,
    float * MC,
    float * alpha,
    float * beta
);
#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)
#define CUSPARSE_SAFE_CALL(func)                                                                \
    do                                                                                          \
    {                                                                                           \
        hipsparseStatus_t e = (func);                                                            \
        if (e != HIPSPARSE_STATUS_SUCCESS)                                                       \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

int cusparse_spmm(
    int M,
    int K,
    int N,
    int *row_index,
    int *col_index,
    float *values,
    int nnz,
    float *MA,
    float *MC,
    float alpha,
    float beta)
{
    /*
    MA: In activation tensor, Shape: M*K
    NOTE: weight need to be transposed if the weight is stores as NxK
    row_index, col_index, values: Weight in CSR format, Shape: K*N
    MC: Output tensor, Shape M*N
    */
    hipsparseHandle_t cusparse_handle;
    printf("M:%d K:%d, N:%d \n", M,K,N);

    CUSPARSE_SAFE_CALL(hipsparseCreate(&cusparse_handle));
    static size_t bufferSize = 0;
    static float *dBuffer = NULL;
    hipsparseSpMatDescr_t sp_weight;
    hipsparseDnMatDescr_t in_activation, output_m;
    printf("M:%d K:%d, N:%d \n", M,K,N);

    // printf("%d\n", row_index[K-1]);
    // printf("%d\n", row_index[K]);
    // int nnz = col_index[row_index[K]];
    printf("nnz:%d\n",nnz);
    CUSPARSE_SAFE_CALL(hipsparseCreateCsr(&sp_weight,
                                         K,
                                         N,
                                         nnz,
                                         (void *)row_index,
                                         (void *)col_index,
                                         (void *)values,
                                         HIPSPARSE_INDEX_32I,
                                         HIPSPARSE_INDEX_32I,
                                         HIPSPARSE_INDEX_BASE_ZERO,
                                         HIP_R_32F));
    CUSPARSE_SAFE_CALL(hipsparseCreateDnMat(&in_activation, K, M, K, MA,
                                           HIP_R_32F, HIPSPARSE_ORDER_COL));
    CUSPARSE_SAFE_CALL(hipsparseCreateDnMat(&output_m, N, M, N, MC,
                                           HIP_R_32F, HIPSPARSE_ORDER_COL));
    if (dBuffer == NULL)
    {
        // allocate the worksparce buffer if this is the first call
        CUSPARSE_SAFE_CALL(hipsparseSpMM_bufferSize(
            cusparse_handle,
            HIPSPARSE_OPERATION_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, sp_weight, in_activation, &beta, output_m, HIP_R_32F,
            HIPSPARSE_SPMM_CSR_ALG2, &bufferSize));
        CUDA_SAFE_CALL(hipMalloc(&dBuffer, bufferSize));
    }
    // Execute the forward matmul
    CUSPARSE_SAFE_CALL(hipsparseSpMM(cusparse_handle,
                                    HIPSPARSE_OPERATION_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, sp_weight, in_activation, &beta, output_m, HIP_R_32F,
                                    HIPSPARSE_SPMM_CSR_ALG2, dBuffer));
    // destroy matrix/vector descriptors
    CUSPARSE_SAFE_CALL(hipsparseDestroyDnMat(in_activation));
    CUSPARSE_SAFE_CALL(hipsparseDestroyDnMat(output_m));
    CUSPARSE_SAFE_CALL(hipsparseDestroySpMat(sp_weight));
    CUSPARSE_SAFE_CALL(hipsparseDestroy(cusparse_handle));
    return 0;
}

at::Tensor cusparse_linear_forward(
    torch::Tensor input,
    torch::Tensor row_index,
    torch::Tensor col_index,
    torch::Tensor values,
    std::vector<int> weight_shape)
{   
    hipSetDevice(input.get_device());
    // the weight shape should be KxN
    int n_dim = input.dim();
    auto input_sizes = input.sizes();
    int in_features = input_sizes[n_dim-1];
    int batch_size = std::accumulate(begin(input_sizes), end(input_sizes), 1, std::multiplies<int>());
    batch_size /= in_features;
    std::vector<int64_t> output_shape;
    for(int i=0; i<n_dim-1; i++) output_shape.push_back(input_sizes[i]);
    assert(weight_shape.size()==2);
    int out_features = weight_shape[1];
    output_shape.push_back(out_features);
    c10::ArrayRef<int64_t> _out_size(output_shape.data(), output_shape.data() + output_shape.size());
    torch::Tensor output = torch::empty(_out_size, input.options());
    printf("row index size: %d\n", row_index.size(0));
    printf("m:%d, k:%d, n:%d\n",batch_size, in_features, out_features);
    int nnz = values.size(0);
    AT_DISPATCH_FLOATING_TYPES(input.type(), "cusparse_linear_forward", ([&]
                                                                           { cusparse_spmm(
                                                                                 batch_size,
                                                                                 in_features,
                                                                                 out_features,
                                                                                 row_index.data_ptr<int>(),
                                                                                 col_index.data_ptr<int>(),
                                                                                 values.data_ptr<float>(),
                                                                                 nnz,
                                                                                 input.data_ptr<float>(),
                                                                                 output.data_ptr<float>(),
                                                                                 1,
                                                                                 0); }));
    return output;
}
