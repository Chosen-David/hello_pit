#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
using namespace std;
// Macro definition for the cuda and cusparse

#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#define OFFSET(row, col, ld) ((row) * ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024
#define FULL_MASK 0xffffffff

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid + 8]; 
    sdata[tid] += sdata[tid + 4]; 
    sdata[tid] += sdata[tid + 2]; 
    sdata[tid] += sdata[tid + 1]; 
}

__device__ __forceinline__ const int* add_ptr_u(const int* src, int offset)      \
{                                                                            \
    const int* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ const float* add_ptr_f(const float* src, int offset)      \
{                                                                            \
    const float* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ float2  _add(float2 x, float2 y) { float2 res; res.x = x.x + y.x; res.y = x.y + y.y; return res; }

__global__ void BLOCK_SPARSE_MATMUL_BIAS_OPENAI(
    float* A,
    float* B,
    float* bias,
    int * index,
    int GLOBAL_M,
    int GLOBAL_K,
    int GLOBAL_N,
    float* output){
    /*
    description:
    tiling k dimension
    smm_dd_s_nt: sparse matmul, dense (MxK, along K) x dense (NxK, along k) -> sparse (MxN, along N)
    the output sparse is block size 32x32, the blocks will be written to bcsr 32x64
    */
    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;

    // A += M * K * blockIdx.z;
    // B += K * N * blockIdx.z;
    // output += M * N * blockIdx.z;
    // int batchid = blockIdx.z;
    // int cur_seq_len = seqlens[batchid];
    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    __shared__ int n_index[BLOCK_SIZE_N];
    __shared__ float bias_share[BLOCK_SIZE_N];
    char* bShare = (char*)fShare;
    uint tid = threadIdx.x;
    uint bx = blockIdx.x; // N
    uint by = blockIdx.y; // M
    if(tid < BLOCK_SIZE_N && bx * BLOCK_SIZE_N + tid < N){
        n_index[tid] = index[bx * BLOCK_SIZE_N + tid];
    }
    uint n_pos;
    if(tid<BLOCK_SIZE_N && bx * BLOCK_SIZE_N + tid < N){
        bias_share[tid] = bias[n_index[tid]]; 
    }
    __syncthreads();

    uint tx = tid % 16;
    uint ty = tid / 16;
    assert(THREAD_SIZE_K % 16 == 0);
    uint k = tx * 4;

    uint ori_offsetA00 = (by * 32 + ty) * K + k;
    uint ori_offsetA16 = ori_offsetA00 + K * 16;
    // uint ori_offsetB00 = (bx * 32 + ty) * K + k;
    // uint ori_offsetB16 = ori_offsetB00 + K * 16;
    // n_pos = bx * 32 + ty;
    uint ori_offsetB00 = n_index[ty] * K + k;
    uint ori_offsetB16 = n_index[ty + 16] * K + k;;

    uint tid224 = tid & 224;
    uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
    uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
    uint loadB = ((tid >> 1) & 7) << 4;
    loadA += (tid224 * 32) + (tid224 / 2);
    loadB += (tid224 * 32) + (tid224 / 2);

    // This keeps all prior logic outside of the loops.
    asm("mov.b32 %0, %0;" : "+r"(storAB) : );
    asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
    asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

    float regC[8][4];
    for (int i = 0; i < 8; i++)
        for (int j = 0; j < 4; j++)
            regC[i][j] = 0.0f;
    // if(threadIdx.x==0){
    //     printf("bx:%d by:%d\n", bx, by);
    // }
    for (int k_seq = 0; k_seq < (int)(K/64); k_seq++)
    {
        // if(k_dim_mask[k_seq]!=1 && tid==0){
        //     printf("k_dim_mask:%d tid:%d bx:%d by:%d\n", k_dim_mask[tid], tid, bx, by);
        // }

        float4 a00 = {0}, a16 = {0};
        float4 b00 = {0}, b16 = {0};
        uint offsetA00 = ori_offsetA00 + 64 * k_seq;
        uint offsetA16 = ori_offsetA16 + 64 * k_seq;
        a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
        a16 = __ldg((const float4*)(add_ptr_f(A, offsetA16)));
        uint offsetB00, offsetB16;
        if(bx * 32 + ty < N){
            offsetB00 = ori_offsetB00 + 64 * k_seq;
            b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
        }
        if(bx*32 + ty+16 < N){
            offsetB16 = ori_offsetB16 + 64 * k_seq;
            b16 = __ldg((const float4*)(add_ptr_f(B, offsetB16)));
        }
        // if(threadIdx.x== 0 && bx == 0 && by == 0){
        //     printf("offsetA00:%d offsetA16:%d offsetB00:%d offsetB16:%d a00:(%f, %f, %f, %f)\n", offsetA00, offsetA16, offsetB00, ori_offsetB16, a00.x, a00.y, a00.z, a00.w);
        // }
        __syncthreads();

        *(float*)&bShare[storAB + (0*32 +  0 + 0*65*32)*4] = a00.x;
        *(float*)&bShare[storAB + (1*32 +  0 + 0*65*32)*4] = a00.y;
        *(float*)&bShare[storAB + (2*32 +  0 + 0*65*32)*4] = a00.z;
        *(float*)&bShare[storAB + (3*32 +  0 + 0*65*32)*4] = a00.w;
        *(float*)&bShare[storAB + (0*32 + 16 + 0*65*32)*4] = a16.x;
        *(float*)&bShare[storAB + (1*32 + 16 + 0*65*32)*4] = a16.y;
        *(float*)&bShare[storAB + (2*32 + 16 + 0*65*32)*4] = a16.z;
        *(float*)&bShare[storAB + (3*32 + 16 + 0*65*32)*4] = a16.w;

        *(float*)&bShare[storAB + (0*32 +  0 + 1*65*32)*4] = b00.x;
        *(float*)&bShare[storAB + (1*32 +  0 + 1*65*32)*4] = b00.y;
        *(float*)&bShare[storAB + (2*32 +  0 + 1*65*32)*4] = b00.z;
        *(float*)&bShare[storAB + (3*32 +  0 + 1*65*32)*4] = b00.w;
        *(float*)&bShare[storAB + (0*32 + 16 + 1*65*32)*4] = b16.x;
        *(float*)&bShare[storAB + (1*32 + 16 + 1*65*32)*4] = b16.y;
        *(float*)&bShare[storAB + (2*32 + 16 + 1*65*32)*4] = b16.z;
        *(float*)&bShare[storAB + (3*32 + 16 + 1*65*32)*4] = b16.w;
        __syncthreads();

        float regA[8], regB[4];
        #pragma unroll
        for (int j = 0; j < 4; j++)
        {
            // fetch outer product data
            *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
            *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
            *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
        #pragma unroll
        for (int j = 4; j < 8; j++)
        {
            *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
            *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
            *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
            *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
            *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
            *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
        
    }

    asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
    asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
    asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

    ty = ((tid & 16) >> 3) + (tid & 1);
    tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

    uint storC = ty*32*8*4 + tx*4;

    tx = tid % 16;
    ty = tid / 16;

    uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);

    // uint blk_index = block_index[blockIdx.x] / 2;
    // uint intra_blk_index = block_index[blockIdx.x] % 2;
    // C_val += 32 * 64 * blk_index + intra_blk_index * 32;
    // C_val += ty * 64 + tx * 2;
    // TODO double check here!
    // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
    //     printf("output offset: %d\n", (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2);
    // }
    float2 re1, re2;
    output += (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2;
    __syncthreads();
    *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
    *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
    *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
    *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
    __syncthreads();

    float2 c2[8];
    for (int i = 0; i < 8; i++)
        c2[i] = *(float2*)&fShare[readC + i*32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i+j]);

    //-> store((bhalf2*)C, c2[0]);
    // *(float2*)C_val = c2[0];

    re1 = _add(c2[0], *(float2*)(bias_share+tx*2));
    // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
    //     printf("output value: %f\n", *output);
    // }

    __syncthreads();
    *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
    *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
    *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
    *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
    __syncthreads();

    for (int i = 0; i < 8; i++)
        c2[i] = *(float2*)&fShare[readC + i*32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i+j]);

    // output += 16 * N;
    // *(float2*)C_val = c2[0];
    re2 = _add(c2[0], *(float2*)(bias_share+tx*2));
    if(blockIdx.x * BLOCK_SIZE_N + tx *2 < N){
        *output = re1.x;
        *(output+16*N) = re2.x;
    }
    if(blockIdx.x * BLOCK_SIZE_N + tx *2 +1 < N){
        *(output+1) = re1.y;
        *(output+16*N+1) = re2.y;
    }
}

__global__ void grad_w_kernel(
    float* A,
    float* B,
    float* C,
    int GLOBAL_M,
    int GLOBAL_K,
    int GLOBAL_N,
    int * index
)
{
    // ori_out_features is on the M dim
    // ori_in_features is on the N dim
    /*
    description:
    tiling k dimension
    smm_dd_s_nn: sparse matmul, dense (MxK, along K) x dense (KxN, along N) -> sparse (MxN, along N)
    the output sparse is block size 32x32, the blocks will be written to bcsr 32x64
    */
    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;

    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    char* bShare = (char*)fShare;

    uint tid = threadIdx.x;
    uint bx = blockIdx.x; // N
    uint by = blockIdx.y; // M

    uint tx = tid % 16;
    uint ty = tid / 16;
    assert(THREAD_SIZE_K % 16 == 0);
    uint k = tx * 4;
    
        // uint ori_offsetA00 = (by * 32 + ty) * K + k;
        // uint ori_offsetA16 = ori_offsetA00 + K * 16;
        // uint ori_offsetB00 = (bx * 32 + ty) * K + k;
        // uint ori_offsetB16 = ori_offsetB00 + K * 16;
        uint ori_offsetA00 = tid / (BLOCK_SIZE_M/4) * M + by * BLOCK_SIZE_M + (tid % (BLOCK_SIZE_M/4)) * 4;
        uint ori_offsetA16 = ori_offsetA00 + M * 32;
        uint ori_offsetB00 = tid / (BLOCK_SIZE_N/4) * N + bx * BLOCK_SIZE_N + (tid % (BLOCK_SIZE_N/4)) * 4;
        uint ori_offsetB16 = ori_offsetB00 + N * 32;

        uint tid224 = tid & 224;
        // uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
        uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
        uint loadB = ((tid >> 1) & 7) << 4;
        uint storA = (tid * 4 + tid / (BLOCK_SIZE_M/4) / 4 *2) * 4;;
        uint storB = (tid * 4 + tid / (BLOCK_SIZE_N/4) / 4 *2) * 4; // (tid *4 + tid / (BLOCK_SIZE_N/4) / 4 * 2)*4

        loadA += (tid224 * 32) + (tid224 / 2);
        loadB += (tid224 * 32) + (tid224 / 2);

        // This keeps all prior logic outside of the loops.
        asm("mov.b32 %0, %0;" : "+r"(storA) : );
        // asm("mov.b32 %0, %0;" : "+r"(storB) : );
        asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
        asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

        float regC[8][4];
        for (int i = 0; i < 8; i++)
        for (int j = 0; j < 4; j++)
        regC[i][j] = 0.0f;

        for (int k_seq = 0; k_seq < (int)(K/64); k_seq++)
        {
        // uint offsetA00 = ori_offsetA00 + 64 * k_seq;
        // uint offsetA16 = ori_offsetA16 + 64 * k_seq;
        // uint offsetB00 = ori_offsetB00 + 64 * k_seq;
        // uint offsetB16 = ori_offsetB16 + 64 * k_seq;
        uint offsetA00 = ori_offsetA00 + 64 * k_seq * M;
        uint offsetA16 = ori_offsetA16 + 64 * k_seq * M;
        uint offsetB00 = ori_offsetB00 + 64 * k_seq * N;
        uint offsetB16 = ori_offsetB16 + 64 * k_seq * N;
        float4 a00 = {0}, a16 = {0};
        float4 b00 = {0}, b16 = {0};
        a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
        a16 = __ldg((const float4*)(add_ptr_f(A, offsetA16)));
        b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
        b16 = __ldg((const float4*)(add_ptr_f(B, offsetB16)));

        __syncthreads();

        // *(float*)&bShare[storAB + (0*32 +  0 + 0*65*32)*4] = a00.x;
        // *(float*)&bShare[storAB + (1*32 +  0 + 0*65*32)*4] = a00.y;
        // *(float*)&bShare[storAB + (2*32 +  0 + 0*65*32)*4] = a00.z;
        // *(float*)&bShare[storAB + (3*32 +  0 + 0*65*32)*4] = a00.w;
        // *(float*)&bShare[storAB + (0*32 + 16 + 0*65*32)*4] = a16.x;
        // *(float*)&bShare[storAB + (1*32 + 16 + 0*65*32)*4] = a16.y;
        // *(float*)&bShare[storAB + (2*32 + 16 + 0*65*32)*4] = a16.z;
        // *(float*)&bShare[storAB + (3*32 + 16 + 0*65*32)*4] = a16.w;

        // *(float*)&bShare[storAB + (0*32 +  0 + 1*65*32)*4] = b00.x;
        // *(float*)&bShare[storAB + (1*32 +  0 + 1*65*32)*4] = b00.y;
        // *(float*)&bShare[storAB + (2*32 +  0 + 1*65*32)*4] = b00.z;
        // *(float*)&bShare[storAB + (3*32 +  0 + 1*65*32)*4] = b00.w;
        // *(float*)&bShare[storAB + (0*32 + 16 + 1*65*32)*4] = b16.x;
        // *(float*)&bShare[storAB + (1*32 + 16 + 1*65*32)*4] = b16.y;
        // *(float*)&bShare[storAB + (2*32 + 16 + 1*65*32)*4] = b16.z;
        // *(float*)&bShare[storAB + (3*32 + 16 + 1*65*32)*4] = b16.w;
        *(float*)&bShare[storA + (0*65*32)*4] = a00.x;
        *(float*)&bShare[storA + (0*65*32 + 1)*4] = a00.y;
        *(float*)&bShare[storA + (0*65*32 + 2)*4] = a00.z;
        *(float*)&bShare[storA + (0*65*32 + 3)*4] = a00.w;
        *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32)*4] = a16.x;
        *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 1)*4] = a16.y;
        *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 2)*4] = a16.z;
        *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 3)*4] = a16.w;

        *(float*)&bShare[storB + (1*65*32)*4] = b00.x;
        *(float*)&bShare[storB + (1*65*32 + 1)*4] = b00.y;
        *(float*)&bShare[storB + (1*65*32 + 2)*4] = b00.z;
        *(float*)&bShare[storB + (1*65*32 + 3)*4] = b00.w;
        *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32)*4] = b16.x;
        *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 1)*4] = b16.y;
        *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 2)*4] = b16.z;
        *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 3)*4] = b16.w;
        __syncthreads();

        float regA[8], regB[4];
        #pragma unroll
        for (int j = 0; j < 4; j++)
        {
        // fetch outer product data
        *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
        *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
        *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

        for (int i = 0; i < 8; i++)
        for (int j = 0; j < 4; j++)
        regC[i][j] += regA[i] * regB[j];
        }
        #pragma unroll
        for (int j = 4; j < 8; j++)
        {
        *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
        *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
        *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
        *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
        *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
        *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

        for (int i = 0; i < 8; i++)
        for (int j = 0; j < 4; j++)
        regC[i][j] += regA[i] * regB[j];
        }
        }

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
        asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

        ty = ((tid & 16) >> 3) + (tid & 1);
        tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

        uint storC = ty*32*8*4 + tx*4;

        tx = tid % 16;
        ty = tid / 16;

        uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);

        // uint blk_index = block_index[blockIdx.x] / 2;
        // uint blk_index = blockIdx.x;
        // uint intra_blk_index = block_index[blockIdx.x] % 2;
        // C_val += 32 * 32 * blk_index;
        // if(threadIdx.x==0 ){
        //     printf("#&& bid:%d blockIdx.y:%d bx:%d by:%d seqlen:%d headid:%d\n", batch_idx, blockIdx.y, (blockIdx.x % (GLOBAL_N / BLOCK_SIZE_N)), (blockIdx.x / (GLOBAL_N / BLOCK_SIZE_N)), cur_seq_len, head_idx);
        // }
        // C_val += ((blockIdx.x / (GLOBAL_N / BLOCK_SIZE_N)) * BLOCK_SIZE_M + ty) * GLOBAL_N + (blockIdx.x % (GLOBAL_N / BLOCK_SIZE_N)) * BLOCK_SIZE_N + tx * 2;
        // // C_val += ty * 32 + tx * 2;
        float * WC;
        WC = C + index[blockIdx.y * BLOCK_SIZE_M + ty] * N + blockIdx.x  * BLOCK_SIZE_N + tx * 2;
        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
        __syncthreads();

        float2 c2[8];
        for (int i = 0; i < 8; i++)
        c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
        c2[i] = _add(c2[i], c2[i+j]);

        //-> store((bhalf2*)C, c2[0]);
        *(float2*)WC = c2[0];

        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
        __syncthreads();

        for (int i = 0; i < 8; i++)
        c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
        c2[i] = _add(c2[i], c2[i+j]);

        // C_val += 16 * 32;
        WC = C + index[blockIdx.y * BLOCK_SIZE_M + ty +16 ] * N + blockIdx.x  * BLOCK_SIZE_N + tx * 2;
        // if(bx==0 && by==0){
        //     printf("tid:%d tx:%d ty:%d M-index:%d c2[0]:(%f %f)\n", threadIdx.x, tx, ty, index[blockIdx.y * BLOCK_SIZE_M + ty +16], c2[0].x, c2[0].y);
        // }
        *(float2*)WC = c2[0];
    

    
}

__global__ void BLOCK_SPARSE_MATMUL_NN_OPENAI(float* A,
                              float* B,
                              float* C,
                              int * index,
                              int GLOBAL_M,
                              int GLOBAL_K,
                              int GLOBAL_N
                              )
{
    /*
    grad_a = grad_c * weight 
            (M * N) * (N * K))
    */
    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;
    // if(blockIdx.x == 0 && blockIdx.y ==0 && threadIdx.x ==0){
    //     printf("M:%d K:%d N:%d\n", M, K, N);
    // }
    // A += M * K * blockIdx.z;
    // B += K * N * blockIdx.z;
    // output += M * N * blockIdx.z;
    // int batchid = blockIdx.z;
    // int cur_seq_len = seqlens[batchid];
    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    __shared__ int k_dim_index[BLOCK_SIZE_K];
    // __shared__ float bias_share[BLOCK_SIZE_N];
    char* bShare = (char*)fShare;

    uint tid = threadIdx.x;
    uint bx = blockIdx.x;
    uint by = blockIdx.y;
    // if(by * BLOCK_SIZE_M < cur_seq_len){
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("by:%d bx:%d bz:%d\n", by, bx, blockIdx.z);
        // }
        // uint bx = n_index[blockIdx.x]; // N
        // uint by = m_index[blockIdx.x]; // M
        // if(tid<BLOCK_SIZE_N){
        //     bias_share[tid] = bias[bx * BLOCK_SIZE_N + tid %32]; 
        // }
        uint tx = tid % 16;
        uint ty = tid / 16;
        assert(THREAD_SIZE_K % 16 == 0);
        uint k = tx * 4;

        uint ori_offsetA00 = (by * 32 + ty) * K + k;
        uint ori_offsetA16 = ori_offsetA00 + K * 16;
        // uint ori_offsetB00 = (bx * 32 + ty) * K + k;
        // uint ori_offsetB16 = ori_offsetB00 + K * 16;
        // K x N -> ori_out_features, ori_in_features
        // uint ori_offsetB00 = tid / (BLOCK_SIZE_N/4) * ori_in_features + bx * BLOCK_SIZE_N + (tid % (BLOCK_SIZE_N/4)) * 4;
        // uint ori_offsetB16 = ori_offsetB00 + ori_in_features * 32;
        uint ori_offsetB00 =  bx * BLOCK_SIZE_N + (tid % (BLOCK_SIZE_N/4)) * 4;
        uint ori_B_k_offset = tid / (BLOCK_SIZE_N/4);

        uint storB = (tid * 4 + tid / (BLOCK_SIZE_N/4) / 4 *2) * 4; // (tid *4 + tid / (BLOCK_SIZE_N/4) / 4 * 2)*4


        uint tid224 = tid & 224;
        uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
        uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
        uint loadB = ((tid >> 1) & 7) << 4;
        loadA += (tid224 * 32) + (tid224 / 2);
        loadB += (tid224 * 32) + (tid224 / 2);

        // This keeps all prior logic outside of the loops.
        asm("mov.b32 %0, %0;" : "+r"(storAB) : );
        asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
        asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

        float regC[8][4];
        for (int i = 0; i < 8; i++)
            for (int j = 0; j < 4; j++)
                regC[i][j] = 0.0f;

        for (int k_seq = 0; k_seq < (int)(K/64); k_seq++)
        {
            if (tid<BLOCK_SIZE_K){
                k_dim_index[tid] = index[tid + 64 * k_seq];
            }
            __syncthreads();
            uint offsetA00 = ori_offsetA00 + 64 * k_seq;
            uint offsetA16 = ori_offsetA16 + 64 * k_seq;
            // uint offsetB00 = ori_offsetB00 + 64 * k_seq;
            // uint offsetB16 = ori_offsetB16 + 64 * k_seq;
            // uint offsetB00 = ori_offsetB00 + 64 * k_seq * ori_in_features;
            // uint offsetB16 = ori_offsetB16 + 64 * k_seq * ori_in_features;
            uint offsetB00 = ori_offsetB00 + k_dim_index[ori_B_k_offset] * N;
            uint offsetB16 = ori_offsetB00 + k_dim_index[ori_B_k_offset + 32] * N;
            float4 a00 = {0}, a16 = {0};
            float4 b00 = {0}, b16 = {0};
            a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
            a16 = __ldg((const float4*)(add_ptr_f(A, offsetA16)));
            b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
            b16 = __ldg((const float4*)(add_ptr_f(B, offsetB16)));

            __syncthreads();

            *(float*)&bShare[storAB + (0*32 +  0 + 0*65*32)*4] = a00.x;
            *(float*)&bShare[storAB + (1*32 +  0 + 0*65*32)*4] = a00.y;
            *(float*)&bShare[storAB + (2*32 +  0 + 0*65*32)*4] = a00.z;
            *(float*)&bShare[storAB + (3*32 +  0 + 0*65*32)*4] = a00.w;
            *(float*)&bShare[storAB + (0*32 + 16 + 0*65*32)*4] = a16.x;
            *(float*)&bShare[storAB + (1*32 + 16 + 0*65*32)*4] = a16.y;
            *(float*)&bShare[storAB + (2*32 + 16 + 0*65*32)*4] = a16.z;
            *(float*)&bShare[storAB + (3*32 + 16 + 0*65*32)*4] = a16.w;


            *(float*)&bShare[storB + (1*65*32)*4] = b00.x;
            *(float*)&bShare[storB + (1*65*32 + 1)*4] = b00.y;
            *(float*)&bShare[storB + (1*65*32 + 2)*4] = b00.z;
            *(float*)&bShare[storB + (1*65*32 + 3)*4] = b00.w;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32)*4] = b16.x;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 1)*4] = b16.y;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 2)*4] = b16.z;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 3)*4] = b16.w;            __syncthreads();

            float regA[8], regB[4];
            #pragma unroll
            for (int j = 0; j < 4; j++)
            {
                // fetch outer product data
                *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
                *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
                *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
            #pragma unroll
            for (int j = 4; j < 8; j++)
            {
                *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
                *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
                *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
                *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
                *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
                *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
        }

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
        asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

        ty = ((tid & 16) >> 3) + (tid & 1);
        tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

        uint storC = ty*32*8*4 + tx*4;

        tx = tid % 16;
        ty = tid / 16;

        uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);

        // uint blk_index = block_index[blockIdx.x] / 2;
        // uint intra_blk_index = block_index[blockIdx.x] % 2;
        // C_val += 32 * 64 * blk_index + intra_blk_index * 32;
        // C_val += ty * 64 + tx * 2;
        // TODO double check here!
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("output offset: %d\n", (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2);
        // }

        C += (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2;
        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
        __syncthreads();

        float2 c2[8];
        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        //-> store((bhalf2*)C, c2[0]);
        // *(float2*)C_val = c2[0];
        *(float2*)C = c2[0];
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("output value: %f\n", *output);
        // }

        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
        __syncthreads();

        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        C += 16 * N;
        // *(float2*)C_val = c2[0];
        *(float2*)C = c2[0];

    // }
    
}

__global__ void BLOCK_SPARSE_MATMUL_NN_BIAS_OPENAI(float* A,
                              float* B,
                              float* bias,
                              int * index,
                              int GLOBAL_M,
                              int GLOBAL_K,
                              int GLOBAL_N,
                              float* C
                              )
{

    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;
    // if(blockIdx.x == 0 && blockIdx.y ==0 && threadIdx.x ==0){
    //     printf("M:%d K:%d N:%d\n", M, K, N);
    // }
    // A += M * K * blockIdx.z;
    // B += K * N * blockIdx.z;
    // output += M * N * blockIdx.z;
    // int batchid = blockIdx.z;
    // int cur_seq_len = seqlens[batchid];
    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    __shared__ int k_dim_index[BLOCK_SIZE_K];
    __shared__ float bias_share[BLOCK_SIZE_N];
    // __shared__ float bias_share[BLOCK_SIZE_N];
    char* bShare = (char*)fShare;

    uint tid = threadIdx.x;
    uint bx = blockIdx.x;
    uint by = blockIdx.y;
    // if(by * BLOCK_SIZE_M < cur_seq_len){
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("by:%d bx:%d bz:%d\n", by, bx, blockIdx.z);
        // }
        // uint bx = n_index[blockIdx.x]; // N
        // uint by = m_index[blockIdx.x]; // M
        if(tid<BLOCK_SIZE_N){
            bias_share[tid] = bias[bx * BLOCK_SIZE_N + tid]; 
        }
        uint tx = tid % 16;
        uint ty = tid / 16;
        assert(THREAD_SIZE_K % 16 == 0);
        uint k = tx * 4;

        uint ori_offsetA00 = (by * 32 + ty) * K + k;
        uint ori_offsetA16 = ori_offsetA00 + K * 16;
        // uint ori_offsetB00 = (bx * 32 + ty) * K + k;
        // uint ori_offsetB16 = ori_offsetB00 + K * 16;
        // K x N -> ori_out_features, ori_in_features
        // uint ori_offsetB00 = tid / (BLOCK_SIZE_N/4) * ori_in_features + bx * BLOCK_SIZE_N + (tid % (BLOCK_SIZE_N/4)) * 4;
        // uint ori_offsetB16 = ori_offsetB00 + ori_in_features * 32;
        uint ori_offsetB00 =  bx * BLOCK_SIZE_N + (tid % (BLOCK_SIZE_N/4)) * 4;
        uint ori_B_k_offset = tid / (BLOCK_SIZE_N/4);

        uint storB = (tid * 4 + tid / (BLOCK_SIZE_N/4) / 4 *2) * 4; // (tid *4 + tid / (BLOCK_SIZE_N/4) / 4 * 2)*4


        uint tid224 = tid & 224;
        uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
        uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
        uint loadB = ((tid >> 1) & 7) << 4;
        loadA += (tid224 * 32) + (tid224 / 2);
        loadB += (tid224 * 32) + (tid224 / 2);

        // This keeps all prior logic outside of the loops.
        asm("mov.b32 %0, %0;" : "+r"(storAB) : );
        asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
        asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

        float regC[8][4];
        for (int i = 0; i < 8; i++)
            for (int j = 0; j < 4; j++)
                regC[i][j] = 0.0f;

        for (int k_seq = 0; k_seq < (int)(K/64); k_seq++)
        {
            if (tid<BLOCK_SIZE_K){
                k_dim_index[tid] = index[tid + 64 * k_seq];
            }
            __syncthreads();
            uint offsetA00 = ori_offsetA00 + 64 * k_seq;
            uint offsetA16 = ori_offsetA16 + 64 * k_seq;
            // uint offsetB00 = ori_offsetB00 + 64 * k_seq;
            // uint offsetB16 = ori_offsetB16 + 64 * k_seq;
            // uint offsetB00 = ori_offsetB00 + 64 * k_seq * ori_in_features;
            // uint offsetB16 = ori_offsetB16 + 64 * k_seq * ori_in_features;
            uint offsetB00 = ori_offsetB00 + k_dim_index[ori_B_k_offset] * N;
            uint offsetB16 = ori_offsetB00 + k_dim_index[ori_B_k_offset + 32] * N;
            float4 a00 = {0}, a16 = {0};
            float4 b00 = {0}, b16 = {0};
            a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
            a16 = __ldg((const float4*)(add_ptr_f(A, offsetA16)));
            b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
            b16 = __ldg((const float4*)(add_ptr_f(B, offsetB16)));
            // if(tid==0 && bx==0 && by==0){
            //     printf("tid:%d offsetB00:%d b00:(%f %f %f %f)\n", tid, offsetB00, b00.x, b00.y, b00.z, b00.w);
            // }
            __syncthreads();

            *(float*)&bShare[storAB + (0*32 +  0 + 0*65*32)*4] = a00.x;
            *(float*)&bShare[storAB + (1*32 +  0 + 0*65*32)*4] = a00.y;
            *(float*)&bShare[storAB + (2*32 +  0 + 0*65*32)*4] = a00.z;
            *(float*)&bShare[storAB + (3*32 +  0 + 0*65*32)*4] = a00.w;
            *(float*)&bShare[storAB + (0*32 + 16 + 0*65*32)*4] = a16.x;
            *(float*)&bShare[storAB + (1*32 + 16 + 0*65*32)*4] = a16.y;
            *(float*)&bShare[storAB + (2*32 + 16 + 0*65*32)*4] = a16.z;
            *(float*)&bShare[storAB + (3*32 + 16 + 0*65*32)*4] = a16.w;


            *(float*)&bShare[storB + (1*65*32)*4] = b00.x;
            *(float*)&bShare[storB + (1*65*32 + 1)*4] = b00.y;
            *(float*)&bShare[storB + (1*65*32 + 2)*4] = b00.z;
            *(float*)&bShare[storB + (1*65*32 + 3)*4] = b00.w;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32)*4] = b16.x;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 1)*4] = b16.y;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 2)*4] = b16.z;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 3)*4] = b16.w;            __syncthreads();

            float regA[8], regB[4];
            #pragma unroll
            for (int j = 0; j < 4; j++)
            {
                // fetch outer product data
                *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
                *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
                *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
            #pragma unroll
            for (int j = 4; j < 8; j++)
            {
                *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
                *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
                *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
                *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
                *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
                *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
        }

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
        asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

        ty = ((tid & 16) >> 3) + (tid & 1);
        tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

        uint storC = ty*32*8*4 + tx*4;

        tx = tid % 16;
        ty = tid / 16;

        uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);

        // uint blk_index = block_index[blockIdx.x] / 2;
        // uint intra_blk_index = block_index[blockIdx.x] % 2;
        // C_val += 32 * 64 * blk_index + intra_blk_index * 32;
        // C_val += ty * 64 + tx * 2;
        // TODO double check here!
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("output offset: %d\n", (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2);
        // }

        C += (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2;
        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
        __syncthreads();

        float2 c2[8];
        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        //-> store((bhalf2*)C, c2[0]);
        // *(float2*)C_val = c2[0];
        // *(float2*)C = c2[0];
        *(float2*)C = _add(c2[0], *(float2*)(bias_share+tx*2));
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("output value: %f\n", *output);
        // }

        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
        __syncthreads();

        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        C += 16 * N;
        // *(float2*)C_val = c2[0];
        // *(float2*)C = c2[0];
        *(float2*)C = _add(c2[0], *(float2*)(bias_share+tx*2));

    // }
    
}


void backward_function( float * activation,
                        float * weight,
                        float * grad_out,
                        int * index,
                        int M,
                        int K,
                        int N,
                        int ori_out_features,
                        float * a_grad,
                        float * w_grad
                        )
{
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 64;
    const int BLOCK_SIZE_N = 32;
    //w_grad(N*K) = grad_c^T(N*M) X activation(M*K)
    dim3 w_block_dim(256);
    dim3 w_grid_dim(K/BLOCK_SIZE_N, N/BLOCK_SIZE_M);
    grad_w_kernel<<<w_grid_dim, w_block_dim>>>(grad_out, activation, w_grad, N, M, K, index);
    //a_grad(M*K) = grad_c(MxN) * weight(NxK)
    dim3 a_grid_dim(K/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    BLOCK_SPARSE_MATMUL_NN_OPENAI<<<a_grid_dim, w_block_dim>>>(grad_out, weight, a_grad, index, M, N, K);
}


void forward_function(  float * activation,
    float* weight,
    float* bias,
    int * index,
    int M,
    int K,
    int N,
    float* output
)
{
    // dense x dense^T -> sparse output
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 64;
    const int BLOCK_SIZE_N = 32;

    dim3 gridDim((N+BLOCK_SIZE_N-1)/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    dim3 blockDim(256);

    BLOCK_SPARSE_MATMUL_BIAS_OPENAI<<<gridDim, blockDim>>>(activation, weight, bias, index, M, K, N, output);

}
void indim_forward_function(  float * activation,
    float* weight,
    float* bias,
    int * index,
    int M,
    int K,
    int N,
    float* output
)
{
    // dense x dense^T -> sparse output
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 64;
    const int BLOCK_SIZE_N = 32;

    dim3 gridDim(N/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    dim3 blockDim(256);

    BLOCK_SPARSE_MATMUL_NN_BIAS_OPENAI<<<gridDim, blockDim>>>(activation, weight, bias, index, M, K, N, output);

}
at::Tensor outdim_dynamic_sparse_linear_forward(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor index,
    torch::Tensor bias
)
{
    hipSetDevice(activation.get_device());   
    int batch_size = activation.size(0);
    int seq_len = activation.size(1);
    int in_hidden = activation.size(2);
    assert(in_hidden==weight.size(1));
    int out_hidden = index.size(0); // NOTE:
    torch::Tensor output = torch::empty({batch_size, seq_len, out_hidden}, activation.options());
    AT_DISPATCH_FLOATING_TYPES(activation.type(), "seqlen_dynamic_sparse_linear", ([&]
        {       forward_function(
                activation.data_ptr<float>(),
                weight.data_ptr<float>(),
                bias.data_ptr<float>(),
                index.data_ptr<int>(),
                batch_size * seq_len,
                in_hidden,
                out_hidden,
                output.data_ptr<float>()
            ); }));
    return output;

}

std::vector<at::Tensor> outdim_dynamic_sparse_linear_backward(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor grad_c,
    torch::Tensor index
)
{
    hipSetDevice(activation.get_device());   
    int batch_size = activation.size(0);
    int seq_len = activation.size(1);
    int in_hidden = activation.size(2);
    assert(in_hidden==weight.size(1));
    int out_hidden = index.size(0); // NOTE:
    int ori_out_hidden = weight.size(0);
    torch::Tensor w_grad = torch::zeros_like(weight);
    torch::Tensor a_grad = torch::empty_like(activation);
    AT_DISPATCH_FLOATING_TYPES(activation.type(), "seqlen_dynamic_sparse_linear", ([&]
    {       backward_function(
            activation.data_ptr<float>(),
            weight.data_ptr<float>(),
            grad_c.data_ptr<float>(),
            index.data_ptr<int>(),
            batch_size * seq_len,
            in_hidden,
            out_hidden,
            ori_out_hidden,
            a_grad.data_ptr<float>(),
            w_grad.data_ptr<float>()
        ); }));
    std::vector<at::Tensor> grads({a_grad, w_grad});
    return grads;
}

at::Tensor indim_dynamic_sparse_linear_forward(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor index,
    torch::Tensor bias
)
{
    hipSetDevice(activation.get_device());   
    int batch_size = activation.size(0);
    int seq_len = activation.size(1);
    int in_hidden = activation.size(2);
    assert(in_hidden==index.size(0));
    int out_hidden = weight.size(1); // NOTE: the weight has been transposed
    torch::Tensor output = torch::empty({batch_size, seq_len, out_hidden}, activation.options());
    AT_DISPATCH_FLOATING_TYPES(activation.type(), "seqlen_dynamic_sparse_linear", ([&]
        {       indim_forward_function(
                activation.data_ptr<float>(),
                weight.data_ptr<float>(),
                bias.data_ptr<float>(),
                index.data_ptr<int>(),
                batch_size * seq_len,
                in_hidden,
                out_hidden,
                output.data_ptr<float>()
            ); }));
    return output;

}

void indim_backward_function( float * activation,
                        float * weight,
                        float * grad_out,
                        int * index,
                        int M,
                        int K,
                        int N,
                        int ori_in_features,
                        float * a_grad,
                        float * w_grad
                        )
{
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 64;
    const int BLOCK_SIZE_N = 32;
    //w_grad(K*N) = activation^T(K*M) * grad_c(M*N)
    dim3 w_block_dim(256);
    // the size of w is K x N
    dim3 w_grid_dim(N/BLOCK_SIZE_N, K/BLOCK_SIZE_M);
    grad_w_kernel<<<w_grid_dim, w_block_dim>>>(activation, grad_out, w_grad, K, M, N, index);
    // //a_grad(M*K) = grad_c(MxN) * weight(NxK)
    // dim3 a_grid_dim(K/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    // BLOCK_SPARSE_MATMUL_NN_OPENAI<<<a_grid_dim, w_block_dim>>>(grad_out, weight, a_grad, index, M, N, K);
}

std::vector<at::Tensor> indim_dynamic_sparse_linear_backward(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor grad_c,
    torch::Tensor index
)
{
    hipSetDevice(activation.get_device());   
    int batch_size = activation.size(0);
    int seq_len = activation.size(1);
    int in_hidden = activation.size(2);
    assert(in_hidden==index.size(0));
    int out_hidden = weight.size(1); // NOTE: the weight has been transposed
    int ori_in_hidden = weight.size(0);
    torch::Tensor w_grad = torch::zeros_like(weight);
    torch::Tensor a_grad = torch::empty_like(activation);
    AT_DISPATCH_FLOATING_TYPES(activation.type(), "seqlen_dynamic_sparse_linear", ([&]
    {       indim_backward_function(
            activation.data_ptr<float>(),
            weight.data_ptr<float>(),
            grad_c.data_ptr<float>(),
            index.data_ptr<int>(),
            batch_size * seq_len,
            in_hidden,
            out_hidden,
            ori_in_hidden,
            a_grad.data_ptr<float>(),
            w_grad.data_ptr<float>()
        ); }));
    std::vector<at::Tensor> grads({a_grad, w_grad});
    return grads;

}