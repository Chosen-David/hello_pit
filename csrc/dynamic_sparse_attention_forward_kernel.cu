#include "hip/hip_runtime.h"

#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
using namespace std;
// Macro definition for the cuda and cusparse

#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#define OFFSET(row, col, ld) ((row) * ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024
#define FULL_MASK 0xffffffff

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid + 8]; 
    sdata[tid] += sdata[tid + 4]; 
    sdata[tid] += sdata[tid + 2]; 
    sdata[tid] += sdata[tid + 1]; 
}

__device__ __forceinline__ const int* add_ptr_u(const int* src, int offset)      \
{                                                                            \
    const int* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ const float* add_ptr_f(const float* src, int offset)      \
{                                                                            \
    const float* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ float2  _add(float2 x, float2 y) { float2 res; res.x = x.x + y.x; res.y = x.y + y.y; return res; }


__global__ void BLOCK_SPARSE_MATMUL_OUT_32_64_32(
    float* A,
    float* B,
    float* C_val,
    int * row_index,
    int * col_index,
    int GLOBAL_M,
    int GLOBAL_K,
    int GLOBAL_N,
    int SPARSE_VAL_SIZE){
    /*
    description:
    tiling k dimension
    smm_dd_s_nn: sparse matmul, dense (MxK, along K) x dense (KxN, along N) -> sparse (MxN, along N)
    the output sparse is block size 32x32, the blocks will be written to bcsr 32x64
    */
    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;

    A += M*K*blockIdx.y;
    B += K*N*blockIdx.y;
    C_val += SPARSE_VAL_SIZE*blockIdx.y;

    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    char* bShare = (char*)fShare;

    uint tid = threadIdx.x;
    uint bx = col_index[blockIdx.x]; // N
    uint by = row_index[blockIdx.x]; // M

    uint tx = tid % 16;
    uint ty = tid / 16;
    assert(THREAD_SIZE_K % 16 == 0);
    uint k = tx * 4;
    uint ori_offsetA00 = (by * 32 + ty) * K + k;
    uint ori_offsetA16 = ori_offsetA00 + K * 16;
    uint ori_offsetB00 = (bx * 32 + ty) * K + k;
    uint ori_offsetB16 = ori_offsetB00 + K * 16;

    uint tid224 = tid & 224;
    uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
    uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
    uint loadB = ((tid >> 1) & 7) << 4;
    loadA += (tid224 * 32) + (tid224 / 2);
    loadB += (tid224 * 32) + (tid224 / 2);

    // This keeps all prior logic outside of the loops.
    asm("mov.b32 %0, %0;" : "+r"(storAB) : );
    asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
    asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

    float regC[8][4];
    for (int i = 0; i < 8; i++)
        for (int j = 0; j < 4; j++)
            regC[i][j] = 0.0f;

    for (int k_seq = 0; k_seq < (int)(K/64); k_seq++)
    {
        uint offsetA00 = ori_offsetA00 + 64 * k_seq;
        uint offsetA16 = ori_offsetA16 + 64 * k_seq;
        uint offsetB00 = ori_offsetB00 + 64 * k_seq;
        uint offsetB16 = ori_offsetB16 + 64 * k_seq;

        float4 a00 = {0}, a16 = {0};
        float4 b00 = {0}, b16 = {0};
        a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
        a16 = __ldg((const float4*)(add_ptr_f(A, offsetA16)));
        b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
        b16 = __ldg((const float4*)(add_ptr_f(B, offsetB16)));

        __syncthreads();

        *(float*)&bShare[storAB + (0*32 +  0 + 0*65*32)*4] = a00.x;
        *(float*)&bShare[storAB + (1*32 +  0 + 0*65*32)*4] = a00.y;
        *(float*)&bShare[storAB + (2*32 +  0 + 0*65*32)*4] = a00.z;
        *(float*)&bShare[storAB + (3*32 +  0 + 0*65*32)*4] = a00.w;
        *(float*)&bShare[storAB + (0*32 + 16 + 0*65*32)*4] = a16.x;
        *(float*)&bShare[storAB + (1*32 + 16 + 0*65*32)*4] = a16.y;
        *(float*)&bShare[storAB + (2*32 + 16 + 0*65*32)*4] = a16.z;
        *(float*)&bShare[storAB + (3*32 + 16 + 0*65*32)*4] = a16.w;

        *(float*)&bShare[storAB + (0*32 +  0 + 1*65*32)*4] = b00.x;
        *(float*)&bShare[storAB + (1*32 +  0 + 1*65*32)*4] = b00.y;
        *(float*)&bShare[storAB + (2*32 +  0 + 1*65*32)*4] = b00.z;
        *(float*)&bShare[storAB + (3*32 +  0 + 1*65*32)*4] = b00.w;
        *(float*)&bShare[storAB + (0*32 + 16 + 1*65*32)*4] = b16.x;
        *(float*)&bShare[storAB + (1*32 + 16 + 1*65*32)*4] = b16.y;
        *(float*)&bShare[storAB + (2*32 + 16 + 1*65*32)*4] = b16.z;
        *(float*)&bShare[storAB + (3*32 + 16 + 1*65*32)*4] = b16.w;
        __syncthreads();

        float regA[8], regB[4];
        #pragma unroll
        for (int j = 0; j < 4; j++)
        {
            // fetch outer product data
            *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
            *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
            *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
        #pragma unroll
        for (int j = 4; j < 8; j++)
        {
            *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
            *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
            *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
            *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
            *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
            *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
    }

    asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
    asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
    asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

    ty = ((tid & 16) >> 3) + (tid & 1);
    tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

    uint storC = ty*32*8*4 + tx*4;

    tx = tid % 16;
    ty = tid / 16;

    uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);

    // uint blk_index = block_index[blockIdx.x] / 2;
    uint blk_index = blockIdx.x;
    // uint intra_blk_index = block_index[blockIdx.x] % 2;
    C_val += 32 * 32 * blk_index;
    C_val += ty * 32 + tx * 2;

    __syncthreads();
    *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
    *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
    *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
    *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
    __syncthreads();

    float2 c2[8];
    for (int i = 0; i < 8; i++)
        c2[i] = *(float2*)&fShare[readC + i*32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i+j]);

    //-> store((bhalf2*)C, c2[0]);
    *(float2*)C_val = c2[0];

    __syncthreads();
    *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
    *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
    *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
    *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
    __syncthreads();

    for (int i = 0; i < 8; i++)
        c2[i] = *(float2*)&fShare[readC + i*32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i+j]);

    C_val += 16 * 32;
    *(float2*)C_val = c2[0];



}
__global__ void SPARSE_SOFTMAX(
    float* C_val,
    float* C_val_mask,
    int* row_index,
    int block_h, int block_w, int SPARSE_VAL_SIZE, int row_tile){
    /*
    description:
    each row of blocks is dealt with a thread group
    each block is 32x32
    */
    C_val += SPARSE_VAL_SIZE*blockIdx.y;

    uint blk_row_idx = blockIdx.x / (block_h/row_tile) ;
    int block_inter_row = (blockIdx.x % (block_h/row_tile)) * row_tile;
    uint bm = threadIdx.x / block_w;
    uint bn = threadIdx.x % block_w;
    assert(block_w % 32==0);
    float regC = 0.0f;
    float regSum = 0.0f;
    int block_seq_start = row_index[blk_row_idx];
    int block_seq_end = row_index[blk_row_idx+1];

    for (int block_seq = block_seq_start; block_seq < block_seq_end; block_seq++) {
        uint index = block_h * block_w * block_seq + (block_inter_row + bm) * block_w + bn;
        // regC = (float)C_val_mask[index]*C_val[index];
        // if (C_val_mask[index] != 0) {
            regC = expf(C_val[index]) * C_val_mask[index];
        // }
        regSum += regC;
    }
    for (int offset = 16; offset > 0; offset /= 2) {
        regSum += __shfl_down_sync(FULL_MASK, regSum, offset);
    }
    regSum = __shfl_sync(FULL_MASK, regSum, 0);
    // if(threadIdx.x%32==1)
    //     printf("Row %d Regsum %f  \n", block_inter_row + bm + blk_row_idx * block_h, regSum);
    for (int block_seq = block_seq_start; block_seq < block_seq_end; block_seq++) {
        uint index = block_h * block_w * block_seq + (block_inter_row + bm) * block_w + bn;
        regC = 0.0f;
        if (C_val_mask[index] > 0) {
            C_val[index] = expf(C_val[index])/regSum;
        }
        else{
            C_val[index] = 0;
        }

    }


}
template <
    const int BLOCK_SIZE_M, // 64
    const int BLOCK_SIZE_K, // 8
    const int BLOCK_SIZE_N, // 128
    const int THREAD_SIZE_M, // 8
    const int THREAD_SIZE_K, // 4
    const int THREAD_SIZE_N  // 8
>
__global__ void BLOCK_SPARSE_MATMUL_SDD(int* csr_row, int * csr_col, float* csr_val, float * B, float* C,  int M, int K, int N, int block_h, int block_w, int sparse_val_size){
    // const int BLOCK_SIZE_M = 32;
    // const int BLOCK_SIZE_K = 32;
    // const int BLOCK_SIZE_N = 64;
    // const int THREAD_SIZE_M = 4;
    // const int THREAD_SIZE_K = 4;
    // const int THREAD_SIZE_N = 4;
    int by = blockIdx.y; // M
    int bx = blockIdx.x; // N
    int bz = blockIdx.z;
    int ty = threadIdx.y; 
    int tx = threadIdx.x;
    csr_val = csr_val + sparse_val_size * bz;
    B = B + K * N * bz;
    C = C + M * N * bz;

    const int padding = 1;
    __shared__ float As[BLOCK_SIZE_M * (padding+BLOCK_SIZE_K)];
    __shared__ float Bs[BLOCK_SIZE_N * (padding+BLOCK_SIZE_K)];

    float accum[THREAD_SIZE_N][THREAD_SIZE_M] = {0};
    float a_frag[THREAD_SIZE_M][THREAD_SIZE_K];
    float b_frag[THREAD_SIZE_N][THREAD_SIZE_K];

    int A_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    int B_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    int bszy = BLOCK_SIZE_M / THREAD_SIZE_M;
    int bszx = BLOCK_SIZE_N / THREAD_SIZE_N;

    int THREADS_PER_BLOCK = bszy * bszx;

    int A_TILE_ROW_STRIDE = THREADS_PER_BLOCK / A_THREAD_PER_ROW;
    int B_TILE_ROW_STRIDE = THREADS_PER_BLOCK / B_THREAD_PER_ROW;

    int tid = ty * bszx + tx;

    int index_start = csr_row[by], index_end = csr_row[by+1];

    int A_BLOCK_ROW_START = tid / A_THREAD_PER_ROW;
    int B_BLOCK_ROW_START = tid / B_THREAD_PER_ROW;

    int A_BLOCK_COL_START = tid % A_THREAD_PER_ROW * 4;
    int B_BLOCK_COL_START = tid % B_THREAD_PER_ROW * 4;
    const int vBLOCK_SIZE_M = BLOCK_SIZE_M / THREAD_SIZE_M;
    const int vBLOCK_SIZE_N = BLOCK_SIZE_N / THREAD_SIZE_N;

    for(int tile_block_idx = index_start; tile_block_idx < index_end; tile_block_idx += 1){
        int col_pos = csr_col[tile_block_idx] * BLOCK_SIZE_K;
        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_M; k += A_TILE_ROW_STRIDE){
            FETCH_FLOAT4(As[OFFSET(k+A_BLOCK_ROW_START, A_BLOCK_COL_START, BLOCK_SIZE_K)]) =
                FETCH_FLOAT4(csr_val[tile_block_idx * BLOCK_SIZE_M * BLOCK_SIZE_K + OFFSET(k+A_BLOCK_ROW_START, A_BLOCK_COL_START, BLOCK_SIZE_K)]);
        }

        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_K; k += B_TILE_ROW_STRIDE){
            FETCH_FLOAT4(Bs[OFFSET(k+B_BLOCK_ROW_START, B_BLOCK_COL_START, BLOCK_SIZE_N)]) = 
                FETCH_FLOAT4(B[OFFSET(col_pos+k+B_BLOCK_ROW_START, bx*BLOCK_SIZE_N + B_BLOCK_COL_START, N)]);
                // FETCH_FLOAT4(W_val[tile_block_idx * BLOCK_SIZE_N * BLOCK_SIZE_K + (k+B_BLOCK_ROW_START) * BLOCK_SIZE_N + B_BLOCK_COL_START]);
                // FETCH_FLOAT4(B[OFFSET(tile_idx+k+B_BLOCK_ROW_START, bx*BLOCK_SIZE_N+B_BLOCK_COL_START, N)]);
        }

        __syncthreads();

        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_K; k += THREAD_SIZE_K){
            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_K; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_M; j += 1){
                    a_frag[j][i] = As[OFFSET(ty + vBLOCK_SIZE_M * j, k+i, BLOCK_SIZE_K)];
                    //a_frag[j][i] = As[OFFSET(k+i, ty + vBLOCK_SIZE_M * j, BLOCK_SIZE_M)];
                }
            }

            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_K; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_N; j += 1){
                    b_frag[j][i] = Bs[OFFSET(k+i, tx + vBLOCK_SIZE_N * j, BLOCK_SIZE_N)];
                }
            }

            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_N; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_M; j++){
                    #pragma unroll
                    for(int k_in = 0; k_in < THREAD_SIZE_K; k_in++){
                        // accum[i][j] = fma(a_frag[j][k_in], b_frag[i][k_in], accum[i][j]);
                        accum[i][j] += a_frag[j][k_in] * b_frag[i][k_in];
                    }
                }
            }
        }

        __syncthreads();
    }


    #pragma unroll
    for(int thread_x = 0; thread_x < THREAD_SIZE_N; thread_x++){
        #pragma unroll
        for(int thread_y = 0; thread_y < THREAD_SIZE_M; thread_y+=1){
            C[OFFSET(
                BLOCK_SIZE_M * by + ty + thread_y * vBLOCK_SIZE_M,
                BLOCK_SIZE_N * bx + tx + thread_x * vBLOCK_SIZE_N,
                N
            )] = (accum[thread_x][thread_y]);
        }
    }


}


void dynamic_forward_function(float* Q, float* K, float* V,
                    float * inter_result, int * row_ptr, int * col_idx, int * row_pos, float * val_mask,
                    int batch_size, int head_num, int seq_length, int hidden_dim, const int block_nnz, float* output)
{
    const int sparse_val_size =  block_nnz * 32* 32 ; //block_nnz * block_h * block_w
    CUDA_SAFE_CALL(hipMemset(inter_result, 0, sizeof(float) * sparse_val_size * batch_size * head_num));
    // already set to zero outside, no need to memset here
    //hipMemset((void*)val, 0, (SPARSE_VAL_SIZE * HEAD_NUM) * batch_size);
    const dim3 dimBlock(256);
    const dim3 dimGrid(block_nnz, head_num * batch_size);
    BLOCK_SPARSE_MATMUL_OUT_32_64_32<<<dimGrid, dimBlock>>>(
        Q,
        K,
        inter_result,
        row_pos,
        col_idx,
        seq_length, // M
        hidden_dim, // K
        seq_length, // N
        sparse_val_size
        
    );

    const int row_tile = 4;
    const dim3 softmax_dimBlock(row_tile*32);
    const dim3 softmax_dimGrid(seq_length/row_tile, head_num * batch_size);
    SPARSE_SOFTMAX<<<softmax_dimGrid, softmax_dimBlock>>>(
        inter_result,
        val_mask,
        row_ptr,
        32,
        32,
        sparse_val_size,
        row_tile);

    // sparse x dense
    // M: seq_length K: seq_length N:hidden dim
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 64;
    const int THREAD_SIZE_M = 4;
    const int THREAD_SIZE_K = 4;
    const int THREAD_SIZE_N = 4;

    dim3 sdd_gridDim(hidden_dim/BLOCK_SIZE_N, seq_length/BLOCK_SIZE_M, head_num * batch_size);
    dim3 sdd_blockDim(BLOCK_SIZE_N/THREAD_SIZE_N, BLOCK_SIZE_M/THREAD_SIZE_M);
    BLOCK_SPARSE_MATMUL_SDD<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_M, THREAD_SIZE_K, THREAD_SIZE_N><<<sdd_gridDim, sdd_blockDim>>>(
        row_ptr,
        col_idx,
        inter_result,
        V,
        output,
        seq_length,
        seq_length,
        hidden_dim,
        32,
        32,
        sparse_val_size);

    
}



at::Tensor dynamic_sparse_attention_forward(
    torch::Tensor Q,
    torch::Tensor K,
    torch::Tensor V,
    torch::Tensor inter_result,
    torch::Tensor row_ptr,
    torch::Tensor col_idx,
    torch::Tensor row_pos,
    torch::Tensor val_mask,
    int block_nnz,
    int head_num
)
{
    hipSetDevice(Q.get_device());
    // Q, K, V should have the same shape which is {batchsize, seq_length, hidden_dim}
    int batch_size = Q.size(0);
    // int head_num = Q.size(1);
    int seq_length = Q.size(2);
    int hidden_dim = Q.size(3);
    torch::Tensor output = torch::empty({batch_size, head_num, seq_length, hidden_dim}, Q.options());
    
    AT_DISPATCH_FLOATING_TYPES(Q.type(), "dynamic_sparse_attention", ([&]
                            { dynamic_forward_function(
                                    Q.data_ptr<float>(),
                                    K.data_ptr<float>(),
                                    V.data_ptr<float>(),
                                    inter_result.data_ptr<float>(),
                                    row_ptr.data_ptr<int>(),
                                    col_idx.data_ptr<int>(),
                                    row_pos.data_ptr<int>(),
                                    val_mask.data_ptr<float>(),
                                    batch_size,
                                    head_num,
                                    seq_length,
                                    hidden_dim,
                                    block_nnz,
                                    output.data_ptr<float>()
                                ); }));
    return output;
}

std::vector<at::Tensor> dynamic_sparse_attention_backward(
    torch::Tensor grad,
    torch::Tensor Q,
    torch::Tensor K,
    torch::Tensor V,
    torch::Tensor gradv_row_idx,
    torch::Tensor gradv_col_idx,
    torch::Tensor gradv_subblock_idx,
    torch::Tensor val,
    torch::Tensor m_index,
    torch::Tensor n_index,
    torch::Tensor block_index,
    torch::Tensor col_range_index,
    torch::Tensor row_ptr,
    torch::Tensor col_idx
    )
{

}
