#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
using namespace std;
// Macro definition for the cuda and cusparse

#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#define OFFSET(row, col, ld) ((row) * ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024
#define FULL_MASK 0xffffffff

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid + 8]; 
    sdata[tid] += sdata[tid + 4]; 
    sdata[tid] += sdata[tid + 2]; 
    sdata[tid] += sdata[tid + 1]; 
}

__device__ __forceinline__ const int* add_ptr_u(const int* src, int offset)      \
{                                                                            \
    const int* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ const float* add_ptr_f(const float* src, int offset)      \
{                                                                            \
    const float* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ float2  _add(float2 x, float2 y) { float2 res; res.x = x.x + y.x; res.y = x.y + y.y; return res; }




__global__ void BLOCK_SPARSE_MATMUL_TN_CONDENSE_OPENAI(float* A, float* B, float*output, int * row_ptr, int * col_idx, int GLOBAL_M, int GLOBAL_K, int GLOBAL_N, int BLOCK_H, int BLOCK_W)
{

    /*
    A : dense matrix with the shape of KxM
    B : dense matrix with the shape of KxN
    */
    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;


    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];

    char* bShare = (char*)fShare;
    uint tid = threadIdx.x;
    uint bx = blockIdx.x; // N
    uint by = blockIdx.y; // M

    __syncthreads();

    uint tx = tid % 16;
    uint ty = tid / 16;
    assert(THREAD_SIZE_K % 16 == 0);
    uint k = tx * 4;

    uint ori_offset_B00 = bx * BLOCK_SIZE_N + (tid % (BLOCK_SIZE_N/4)) * 4;
    uint ori_offset_A00 = by * BLOCK_SIZE_M + (tid % (BLOCK_SIZE_M/4)) * 4;
    uint storB = (tid * 4 + tid / (BLOCK_SIZE_N/4) / 4 *2) * 4; // (tid *4 + tid / (BLOCK_SIZE_N/4) / 4 * 2)*4
    uint storA = (tid * 4 + tid / (BLOCK_SIZE_M/4) / 4 *2) * 4; // (tid *4 + tid / (BLOCK_SIZE_N/4) / 4 * 2)*4
    uint tid224 = tid & 224;
    uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
    uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
    uint loadB = ((tid >> 1) & 7) << 4;
    loadA += (tid224 * 32) + (tid224 / 2);
    loadB += (tid224 * 32) + (tid224 / 2);

    // This keeps all prior logic outside of the loops.
    asm("mov.b32 %0, %0;" : "+r"(storAB) : );
    asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
    asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

    float regC[8][4];
    for (int i = 0; i < 8; i++)
        for (int j = 0; j < 4; j++)
            regC[i][j] = 0.0f;
    int index_start = row_ptr[bx], index_end = row_ptr[bx+1];
    // float4 const0 = {0};
    int round = (index_end - index_start - 1 + BLOCK_SIZE_K/BLOCK_H) / (BLOCK_SIZE_K/BLOCK_H);
    for (int rid = 0; rid<round; rid++)
    {
        uint k_offset = tid / (BLOCK_SIZE_N/4) + rid * BLOCK_SIZE_K;
        uint k_offset32 = k_offset + 32;
        // offsetA00 = offsetA00 + BLOCK_SIZE_K * BLOCK_SIZE_M; 
        // offsetA32 = offsetA32 + BLOCK_SIZE_K * BLOCK_SIZE_M;
        uint _pos = (k_offset / BLOCK_H);
        uint _pos32 = (k_offset32/BLOCK_H);
        uint offsetB00 = (col_idx[index_start+_pos]+k_offset%BLOCK_H) * N + ori_offset_B00;
        uint offsetB32 = (col_idx[index_start+_pos32]+k_offset32%BLOCK_H) * N + ori_offset_B00;
        uint offsetA00 = (col_idx[index_start+_pos]+k_offset%BLOCK_H) * M + ori_offset_A00;;
        uint offsetA32 = (col_idx[index_start+_pos32]+k_offset32%BLOCK_H) * M + ori_offset_A00;
        // uint offsetB00 = ori_offsetB00 + 64 * A_col[bcsr_col_idx] * N;
        // uint offsetB16 = ori_offsetB16 + 64 * A_col[bcsr_col_idx] * N;
        // if(bx==0 && by == 0 &&  threadIdx.x==0){
        //     printf("_pos:%d _pos32:%d index_end-index_start:%d\n", _pos, _pos32, index_end-index_start);
        // }
        float4 a00 = {0,0,0,0}, a16 = {0,0,0,0};
        float4 b00 = {0,0,0,0}, b16 = {0,0,0,0};
        if(_pos<index_end-index_start){
            a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
            b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
        }
        if(_pos32<index_end-index_start){
            a16 = __ldg((const float4*)(add_ptr_f(A, offsetA32)));
            b16 = __ldg((const float4*)(add_ptr_f(B, offsetB32)));
        }

        __syncthreads();

        *(float*)&bShare[storA + (0*65*32)*4] = a00.x;
        *(float*)&bShare[storA + (0*65*32 + 1)*4] = a00.y;
        *(float*)&bShare[storA + (0*65*32 + 2)*4] = a00.z;
        *(float*)&bShare[storA + (0*65*32 + 3)*4] = a00.w;
        *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32)*4] = a16.x;
        *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 1)*4] = a16.y;
        *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 2)*4] = a16.z;
        *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 3)*4] = a16.w;

        *(float*)&bShare[storB + (1*65*32)*4] = b00.x;
        *(float*)&bShare[storB + (1*65*32 + 1)*4] = b00.y;
        *(float*)&bShare[storB + (1*65*32 + 2)*4] = b00.z;
        *(float*)&bShare[storB + (1*65*32 + 3)*4] = b00.w;
        *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32)*4] = b16.x;
        *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 1)*4] = b16.y;
        *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 2)*4] = b16.z;
        *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 3)*4] = b16.w;

        __syncthreads();

        float regA[8], regB[4];
        #pragma unroll
        for (int j = 0; j < 4; j++)
        {
            // fetch outer product data
            *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
            *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
            *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
        #pragma unroll
        for (int j = 4; j < 8; j++)
        {
            *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
            *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
            *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
            *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
            *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
            *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
    
    }

    asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
    asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
    asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

    ty = ((tid & 16) >> 3) + (tid & 1);
    tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

    uint storC = ty*32*8*4 + tx*4;

    tx = tid % 16;
    ty = tid / 16;

    uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);


    output += (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2;
    __syncthreads();
    *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
    *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
    *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
    *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
    __syncthreads();

    float2 c2[8];
    for (int i = 0; i < 8; i++)
        c2[i] = *(float2*)&fShare[readC + i*32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i+j]);

    //-> store((bhalf2*)C, c2[0]);
    // *(float2*)C_val = c2[0];
    *(float2*)output = c2[0];
    // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
    //     printf("output value: %f\n", *output);
    // }

    __syncthreads();
    *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
    *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
    *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
    *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
    __syncthreads();

    for (int i = 0; i < 8; i++)
        c2[i] = *(float2*)&fShare[readC + i*32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i+j]);

    output += 16 * N;
    // *(float2*)C_val = c2[0];
    *(float2*)output = c2[0];

}

void condense_dynamic_forward_function(float* activation, float* weight, int* row_ptr, int* col_idx,
                    float* bias, int M, int K, int N, int block_h, int block_w, float* output)
{
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 64;
    const int BLOCK_SIZE_N = 32;
    dim3 gridDim(N/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    dim3 blockDim(256);
    BLOCK_SPARSE_MATMUL_TN_CONDENSE_OPENAI<<<gridDim, blockDim>>>(activation, weight, output, row_ptr, col_idx, M, K, N, block_h, block_w);
}

void dynamic_backward_function(float* grad_in, int * row_ptr, int *col_idx, float* val, int M, int K, int N, int block_h, int block_w, float* grad_out)
{

}

at::Tensor dynamic_sparse_linear_condense_forward(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor row_ptr,
    torch::Tensor col_index,
    torch::Tensor bias,
    int M, int K, int N, int block_h, int block_w
)
{
    // The weight tensor here should be transposed and in the shape of K x N
    hipSetDevice(activation.get_device());
    int batch_size = activation.size(0);
    int seq_len = activation.size(1);
    int in_hidden = activation.size(2);
    assert(in_hidden == weight.size(0));
    assert(M == batch_size* seq_len);
    int out_hidden = weight.size(1);
    torch::Tensor output = torch::empty({batch_size, seq_len, out_hidden}, activation.options());
    // Q, K, V should have the same shape which is {batchsize, seq_length, hidden_dim}

    
    AT_DISPATCH_FLOATING_TYPES(activation.type(), "dynamic_sparse_linear", ([&]
                            { condense_dynamic_forward_function(
                                    activation.data_ptr<float>(),
                                    weight.data_ptr<float>(),
                                    row_ptr.data_ptr<int>(),
                                    col_index.data_ptr<int>(),
                                    bias.data_ptr<float>(),
                                    M, K, N, block_h, block_w,
                                    output.data_ptr<float>()
                                ); }));
    return output;
}

vector<at::Tensor> dynamic_sparse_linear_condense_backward(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor grad_a_row_ptr,
    torch::Tensor grad_a_col_index,
    torch::Tensor grad_c,
    int M, int K, int N, int block_h, int block_w
)
{
    vector<at::Tensor> grads;
    return grads;
}