#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
using namespace std;
// Macro definition for the cuda and cusparse

#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#define OFFSET(row, col, ld) ((row) * ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024
#define FULL_MASK 0xffffffff

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid + 8]; 
    sdata[tid] += sdata[tid + 4]; 
    sdata[tid] += sdata[tid + 2]; 
    sdata[tid] += sdata[tid + 1]; 
}

__device__ __forceinline__ const int* add_ptr_u(const int* src, int offset)      \
{                                                                            \
    const int* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ const float* add_ptr_f(const float* src, int offset)      \
{                                                                            \
    const float* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ float2  _add(float2 x, float2 y) { float2 res; res.x = x.x + y.x; res.y = x.y + y.y; return res; }



__global__ void BLOCK_SPARSE_MATMUL_BIAS_OPENAI(
    float* A,
    float* B,
    float* bias,
    int ori_in_features,
    int ori_out_features,
    int GLOBAL_M,
    int GLOBAL_K,
    int GLOBAL_N,
    int batchsize,
    float* output){
    /*
    description:
    tiling k dimension
    smm_dd_s_nt: sparse matmul, dense (MxK, along K) x dense (NxK, along k) -> sparse (MxN, along N)
    the output sparse is block size 32x32, the blocks will be written to bcsr 32x64
    */
    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;

    // A += M * K * blockIdx.z;
    // B += K * N * blockIdx.z;
    // output += M * N * blockIdx.z;
    // int batchid = blockIdx.z;
    // int cur_seq_len = seqlens[batchid];
    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    __shared__ float bias_share[BLOCK_SIZE_N];
    char* bShare = (char*)fShare;

    uint tid = threadIdx.x;
    uint bx = blockIdx.x;
    uint by = blockIdx.y;
    // if(by * BLOCK_SIZE_M < cur_seq_len){
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("by:%d bx:%d bz:%d\n", by, bx, blockIdx.z);
        // }
        // uint bx = n_index[blockIdx.x]; // N
        // uint by = m_index[blockIdx.x]; // M
        if(tid<BLOCK_SIZE_N){
            bias_share[tid] = bias[bx * BLOCK_SIZE_N + tid %32]; 
        }
        uint tx = tid % 16;
        uint ty = tid / 16;
        assert(THREAD_SIZE_K % 16 == 0);
        uint k = tx * 4;

        uint ori_offsetA00 = (by * 32 + ty) * K + k;
        uint ori_offsetA16 = ori_offsetA00 + K * 16;
        uint ori_offsetB00 = (bx * 32 + ty) * ori_in_features + k;
        uint ori_offsetB16 = ori_offsetB00 + ori_in_features * 16;

        uint tid224 = tid & 224;
        uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
        uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
        uint loadB = ((tid >> 1) & 7) << 4;
        loadA += (tid224 * 32) + (tid224 / 2);
        loadB += (tid224 * 32) + (tid224 / 2);

        // This keeps all prior logic outside of the loops.
        asm("mov.b32 %0, %0;" : "+r"(storAB) : );
        asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
        asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

        float regC[8][4];
        for (int i = 0; i < 8; i++)
            for (int j = 0; j < 4; j++)
                regC[i][j] = 0.0f;

        for (int k_seq = 0; k_seq < (int)(K/64); k_seq++)
        {
            uint offsetA00 = ori_offsetA00 + 64 * k_seq;
            uint offsetA16 = ori_offsetA16 + 64 * k_seq;
            uint offsetB00 = ori_offsetB00 + 64 * k_seq;
            uint offsetB16 = ori_offsetB16 + 64 * k_seq;

            float4 a00 = {0}, a16 = {0};
            float4 b00 = {0}, b16 = {0};
            a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
            a16 = __ldg((const float4*)(add_ptr_f(A, offsetA16)));
            b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
            b16 = __ldg((const float4*)(add_ptr_f(B, offsetB16)));

            __syncthreads();

            *(float*)&bShare[storAB + (0*32 +  0 + 0*65*32)*4] = a00.x;
            *(float*)&bShare[storAB + (1*32 +  0 + 0*65*32)*4] = a00.y;
            *(float*)&bShare[storAB + (2*32 +  0 + 0*65*32)*4] = a00.z;
            *(float*)&bShare[storAB + (3*32 +  0 + 0*65*32)*4] = a00.w;
            *(float*)&bShare[storAB + (0*32 + 16 + 0*65*32)*4] = a16.x;
            *(float*)&bShare[storAB + (1*32 + 16 + 0*65*32)*4] = a16.y;
            *(float*)&bShare[storAB + (2*32 + 16 + 0*65*32)*4] = a16.z;
            *(float*)&bShare[storAB + (3*32 + 16 + 0*65*32)*4] = a16.w;

            *(float*)&bShare[storAB + (0*32 +  0 + 1*65*32)*4] = b00.x;
            *(float*)&bShare[storAB + (1*32 +  0 + 1*65*32)*4] = b00.y;
            *(float*)&bShare[storAB + (2*32 +  0 + 1*65*32)*4] = b00.z;
            *(float*)&bShare[storAB + (3*32 +  0 + 1*65*32)*4] = b00.w;
            *(float*)&bShare[storAB + (0*32 + 16 + 1*65*32)*4] = b16.x;
            *(float*)&bShare[storAB + (1*32 + 16 + 1*65*32)*4] = b16.y;
            *(float*)&bShare[storAB + (2*32 + 16 + 1*65*32)*4] = b16.z;
            *(float*)&bShare[storAB + (3*32 + 16 + 1*65*32)*4] = b16.w;
            __syncthreads();

            float regA[8], regB[4];
            #pragma unroll
            for (int j = 0; j < 4; j++)
            {
                // fetch outer product data
                *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
                *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
                *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
            #pragma unroll
            for (int j = 4; j < 8; j++)
            {
                *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
                *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
                *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
                *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
                *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
                *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
        }

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
        asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

        ty = ((tid & 16) >> 3) + (tid & 1);
        tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

        uint storC = ty*32*8*4 + tx*4;

        tx = tid % 16;
        ty = tid / 16;

        uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);

        // uint blk_index = block_index[blockIdx.x] / 2;
        // uint intra_blk_index = block_index[blockIdx.x] % 2;
        // C_val += 32 * 64 * blk_index + intra_blk_index * 32;
        // C_val += ty * 64 + tx * 2;
        // TODO double check here!
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("output offset: %d\n", (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2);
        // }

        output += (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2;
        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
        __syncthreads();

        float2 c2[8];
        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        //-> store((bhalf2*)C, c2[0]);
        // *(float2*)C_val = c2[0];
        *(float2*)output = _add(c2[0], *(float2*)(bias_share+tx*2));
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("output value: %f\n", *output);
        // }

        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
        __syncthreads();

        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        output += 16 * N;
        // *(float2*)C_val = c2[0];
        *(float2*)output = _add(c2[0], *(float2*)(bias_share+tx*2));

    // }
}



void elastic_forward_function(float* activation,
                                float* weight,
                                float * bias,
                                int ori_in_features,
                                int ori_out_features,
                                int M,
                                int K,
                                int N,
                                int batchsize,
                                float*output)
{

    // dense x dense^T -> sparse output
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 64;
    const int BLOCK_SIZE_N = 32;
    
    dim3 gridDim(N/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    dim3 blockDim(256);
    // printf("gridDim: %d %d %d")
    BLOCK_SPARSE_MATMUL_BIAS_OPENAI<<<gridDim, blockDim>>>(activation, weight, bias, ori_in_features, ori_out_features, M, K, N, batchsize, output);
    
}


at::Tensor elastic_sparse_linear_forward(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor bias,
    int in_features,
    int out_features
)
{
    hipSetDevice(activation.get_device());
    int batch_size = activation.size(0);
    int max_seq_len = activation.size(1);
    int ori_in_features = weight.size(1);
    int ori_out_features = weight.size(0);
    int M = max_seq_len * batch_size;
    int K = in_features;
    int N = out_features;
    assert(in_features % 64==0);
    assert(out_features % 32==0);
    // Q, K, V should have the same shape which is {batchsize, seq_length, hidden_dim}
    torch::Tensor output = torch::empty({batch_size, max_seq_len, out_features}, activation.options());
    
    AT_DISPATCH_FLOATING_TYPES(activation.type(), "seqlen_dynamic_sparse_linear", ([&]
                            {       elastic_forward_function(
                                    activation.data_ptr<float>(),
                                    weight.data_ptr<float>(),
                                    bias.data_ptr<float>(),
                                    ori_in_features,
                                    ori_out_features,
                                    M, K, N, batch_size,
                                    output.data_ptr<float>()
                                ); }));
    return output;
}

__global__ void grad_w_kernel(float* A,
                              float* B,
                              float* C,
                              int GLOBAL_M,
                              int GLOBAL_K,
                              int GLOBAL_N,
                              int ori_in_feature,
                              int ori_out_feature)
{
    // ori_out_features is on the M dim
    // ori_in_features is on the N dim
    /*
    description:
    tiling k dimension
    smm_dd_s_nn: sparse matmul, dense (MxK, along K) x dense (KxN, along N) -> sparse (MxN, along N)
    the output sparse is block size 32x32, the blocks will be written to bcsr 32x64
    */
    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;
    // int batch_idx = blockIdx.z;
    // int head_idx = blockIdx.y + gridDim.y * blockIdx.z; // launch config: block_idx, head_idx, batch_idx
    // if(threadIdx.x==0 && blockIdx.x==0){
    //     printf("hid:%d, bY:%d, bZ:%d , gdim:%d \n", head_idx, blockIdx.y, blockIdx.z, gridDim.y);
    // }
    // A += M * K * head_idx;
    // B += K * N * head_idx;
    // C_val += GLOBAL_M * GLOBAL_N * head_idx;
    // uint cur_seq_len = seqlens[batch_idx];

    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    char* bShare = (char*)fShare;

    uint tid = threadIdx.x;
    uint bx = blockIdx.x; // N
    uint by = blockIdx.y; // M
    // uint bx = (blockIdx.x % (GLOBAL_N / BLOCK_SIZE_N));
    // uint by = (blockIdx.x / (GLOBAL_N / BLOCK_SIZE_N));
    // uint bx = col_index[blockIdx.x]; // N
    // uint by = row_index[blockIdx.x]; // M

    // if (bx * BLOCK_SIZE_N < cur_seq_len && by * BLOCK_SIZE_M < cur_seq_len){
        // if(threadIdx.x==0 ){
        //     printf("## bid:%d blockIdx.y:%d bx:%d by:%d seqlen:%d headid:%d\n", batch_idx, blockIdx.y, bx, by, cur_seq_len, head_idx);
        // }
        uint tx = tid % 16;
        uint ty = tid / 16;
        assert(THREAD_SIZE_K % 16 == 0);
        uint k = tx * 4;
        // uint ori_offsetA00 = (by * 32 + ty) * K + k;
        // uint ori_offsetA16 = ori_offsetA00 + K * 16;
        // uint ori_offsetB00 = (bx * 32 + ty) * K + k;
        // uint ori_offsetB16 = ori_offsetB00 + K * 16;
        uint ori_offsetA00 = tid / (BLOCK_SIZE_M/4) * M + by * BLOCK_SIZE_M + (tid % (BLOCK_SIZE_M/4)) * 4;
        uint ori_offsetA16 = ori_offsetA00 + M * 32;
        uint ori_offsetB00 = tid / (BLOCK_SIZE_N/4) * N + bx * BLOCK_SIZE_N + (tid % (BLOCK_SIZE_N/4)) * 4;
        uint ori_offsetB16 = ori_offsetB00 + N * 32;

        uint tid224 = tid & 224;
        // uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
        uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
        uint loadB = ((tid >> 1) & 7) << 4;
        uint storA = (tid * 4 + tid / (BLOCK_SIZE_M/4) / 4 *2) * 4;;
        uint storB = (tid * 4 + tid / (BLOCK_SIZE_N/4) / 4 *2) * 4; // (tid *4 + tid / (BLOCK_SIZE_N/4) / 4 * 2)*4

        loadA += (tid224 * 32) + (tid224 / 2);
        loadB += (tid224 * 32) + (tid224 / 2);

        // This keeps all prior logic outside of the loops.
        asm("mov.b32 %0, %0;" : "+r"(storA) : );
        // asm("mov.b32 %0, %0;" : "+r"(storB) : );
        asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
        asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

        float regC[8][4];
        for (int i = 0; i < 8; i++)
            for (int j = 0; j < 4; j++)
                regC[i][j] = 0.0f;

        for (int k_seq = 0; k_seq < (int)(K/64); k_seq++)
        {
            // uint offsetA00 = ori_offsetA00 + 64 * k_seq;
            // uint offsetA16 = ori_offsetA16 + 64 * k_seq;
            // uint offsetB00 = ori_offsetB00 + 64 * k_seq;
            // uint offsetB16 = ori_offsetB16 + 64 * k_seq;
            uint offsetA00 = ori_offsetA00 + 64 * k_seq * M;
            uint offsetA16 = ori_offsetA16 + 64 * k_seq * M;
            uint offsetB00 = ori_offsetB00 + 64 * k_seq * N;
            uint offsetB16 = ori_offsetB16 + 64 * k_seq * N;
            float4 a00 = {0}, a16 = {0};
            float4 b00 = {0}, b16 = {0};
            a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
            a16 = __ldg((const float4*)(add_ptr_f(A, offsetA16)));
            b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
            b16 = __ldg((const float4*)(add_ptr_f(B, offsetB16)));

            __syncthreads();

            // *(float*)&bShare[storAB + (0*32 +  0 + 0*65*32)*4] = a00.x;
            // *(float*)&bShare[storAB + (1*32 +  0 + 0*65*32)*4] = a00.y;
            // *(float*)&bShare[storAB + (2*32 +  0 + 0*65*32)*4] = a00.z;
            // *(float*)&bShare[storAB + (3*32 +  0 + 0*65*32)*4] = a00.w;
            // *(float*)&bShare[storAB + (0*32 + 16 + 0*65*32)*4] = a16.x;
            // *(float*)&bShare[storAB + (1*32 + 16 + 0*65*32)*4] = a16.y;
            // *(float*)&bShare[storAB + (2*32 + 16 + 0*65*32)*4] = a16.z;
            // *(float*)&bShare[storAB + (3*32 + 16 + 0*65*32)*4] = a16.w;

            // *(float*)&bShare[storAB + (0*32 +  0 + 1*65*32)*4] = b00.x;
            // *(float*)&bShare[storAB + (1*32 +  0 + 1*65*32)*4] = b00.y;
            // *(float*)&bShare[storAB + (2*32 +  0 + 1*65*32)*4] = b00.z;
            // *(float*)&bShare[storAB + (3*32 +  0 + 1*65*32)*4] = b00.w;
            // *(float*)&bShare[storAB + (0*32 + 16 + 1*65*32)*4] = b16.x;
            // *(float*)&bShare[storAB + (1*32 + 16 + 1*65*32)*4] = b16.y;
            // *(float*)&bShare[storAB + (2*32 + 16 + 1*65*32)*4] = b16.z;
            // *(float*)&bShare[storAB + (3*32 + 16 + 1*65*32)*4] = b16.w;
            *(float*)&bShare[storA + (0*65*32)*4] = a00.x;
            *(float*)&bShare[storA + (0*65*32 + 1)*4] = a00.y;
            *(float*)&bShare[storA + (0*65*32 + 2)*4] = a00.z;
            *(float*)&bShare[storA + (0*65*32 + 3)*4] = a00.w;
            *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32)*4] = a16.x;
            *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 1)*4] = a16.y;
            *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 2)*4] = a16.z;
            *(float*)&bShare[storA + (32*32 + 8*2 + 0*65*32 + 3)*4] = a16.w;

            *(float*)&bShare[storB + (1*65*32)*4] = b00.x;
            *(float*)&bShare[storB + (1*65*32 + 1)*4] = b00.y;
            *(float*)&bShare[storB + (1*65*32 + 2)*4] = b00.z;
            *(float*)&bShare[storB + (1*65*32 + 3)*4] = b00.w;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32)*4] = b16.x;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 1)*4] = b16.y;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 2)*4] = b16.z;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 3)*4] = b16.w;
            __syncthreads();

            float regA[8], regB[4];
            #pragma unroll
            for (int j = 0; j < 4; j++)
            {
                // fetch outer product data
                *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
                *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
                *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
            #pragma unroll
            for (int j = 4; j < 8; j++)
            {
                *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
                *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
                *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
                *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
                *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
                *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
        }

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
        asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

        ty = ((tid & 16) >> 3) + (tid & 1);
        tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

        uint storC = ty*32*8*4 + tx*4;

        tx = tid % 16;
        ty = tid / 16;

        uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);

        // uint blk_index = block_index[blockIdx.x] / 2;
        // uint blk_index = blockIdx.x;
        // uint intra_blk_index = block_index[blockIdx.x] % 2;
        // C_val += 32 * 32 * blk_index;
        // if(threadIdx.x==0 ){
        //     printf("#&& bid:%d blockIdx.y:%d bx:%d by:%d seqlen:%d headid:%d\n", batch_idx, blockIdx.y, (blockIdx.x % (GLOBAL_N / BLOCK_SIZE_N)), (blockIdx.x / (GLOBAL_N / BLOCK_SIZE_N)), cur_seq_len, head_idx);
        // }
        // C_val += ((blockIdx.x / (GLOBAL_N / BLOCK_SIZE_N)) * BLOCK_SIZE_M + ty) * GLOBAL_N + (blockIdx.x % (GLOBAL_N / BLOCK_SIZE_N)) * BLOCK_SIZE_N + tx * 2;
        // // C_val += ty * 32 + tx * 2;
        C += (blockIdx.y * BLOCK_SIZE_M + ty) * ori_in_feature + blockIdx.x  * BLOCK_SIZE_N + tx * 2;
        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
        __syncthreads();

        float2 c2[8];
        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        //-> store((bhalf2*)C, c2[0]);
        *(float2*)C = c2[0];

        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
        __syncthreads();

        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        // C_val += 16 * 32;
        C += 16 * GLOBAL_N;
        *(float2*)C = c2[0];


    // }
}

__global__ void grad_a_kernel(float* A,
                              float* B,
                              float* C,
                              int GLOBAL_M,
                              int GLOBAL_K,
                              int GLOBAL_N,
                              int ori_in_features,
                              int ori_out_featuress
                              )
{
    /*
    grad_a = grad_c * weight 
            (M * N) * (N * K))
    */
    const int BLOCK_SIZE_M = 32;  // 64
    const int BLOCK_SIZE_K = 64;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;

    // A += M * K * blockIdx.z;
    // B += K * N * blockIdx.z;
    // output += M * N * blockIdx.z;
    // int batchid = blockIdx.z;
    // int cur_seq_len = seqlens[batchid];
    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    // __shared__ float bias_share[BLOCK_SIZE_N];
    char* bShare = (char*)fShare;

    uint tid = threadIdx.x;
    uint bx = blockIdx.x;
    uint by = blockIdx.y;
    // if(by * BLOCK_SIZE_M < cur_seq_len){
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("by:%d bx:%d bz:%d\n", by, bx, blockIdx.z);
        // }
        // uint bx = n_index[blockIdx.x]; // N
        // uint by = m_index[blockIdx.x]; // M
        // if(tid<BLOCK_SIZE_N){
        //     bias_share[tid] = bias[bx * BLOCK_SIZE_N + tid %32]; 
        // }
        uint tx = tid % 16;
        uint ty = tid / 16;
        assert(THREAD_SIZE_K % 16 == 0);
        uint k = tx * 4;

        uint ori_offsetA00 = (by * 32 + ty) * K + k;
        uint ori_offsetA16 = ori_offsetA00 + K * 16;
        // uint ori_offsetB00 = (bx * 32 + ty) * K + k;
        // uint ori_offsetB16 = ori_offsetB00 + K * 16;
        // K x N -> ori_out_features, ori_in_features
        uint ori_offsetB00 = tid / (BLOCK_SIZE_N/4) * ori_in_features + bx * BLOCK_SIZE_N + (tid % (BLOCK_SIZE_N/4)) * 4;
        uint ori_offsetB16 = ori_offsetB00 + N * 32;
        uint storB = (tid * 4 + tid / (BLOCK_SIZE_N/4) / 4 *2) * 4; // (tid *4 + tid / (BLOCK_SIZE_N/4) / 4 * 2)*4


        uint tid224 = tid & 224;
        uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
        uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
        uint loadB = ((tid >> 1) & 7) << 4;
        loadA += (tid224 * 32) + (tid224 / 2);
        loadB += (tid224 * 32) + (tid224 / 2);

        // This keeps all prior logic outside of the loops.
        asm("mov.b32 %0, %0;" : "+r"(storAB) : );
        asm("mov.b32 %0, %0;" : "+r"(loadA)  : );
        asm("mov.b32 %0, %0;" : "+r"(loadB)  : );

        float regC[8][4];
        for (int i = 0; i < 8; i++)
            for (int j = 0; j < 4; j++)
                regC[i][j] = 0.0f;

        for (int k_seq = 0; k_seq < (int)(K/64); k_seq++)
        {
            uint offsetA00 = ori_offsetA00 + 64 * k_seq;
            uint offsetA16 = ori_offsetA16 + 64 * k_seq;
            // uint offsetB00 = ori_offsetB00 + 64 * k_seq;
            // uint offsetB16 = ori_offsetB16 + 64 * k_seq;
            uint offsetB00 = ori_offsetB00 + 64 * k_seq * ori_in_features;
            uint offsetB16 = ori_offsetB16 + 64 * k_seq * ori_in_features;

            float4 a00 = {0}, a16 = {0};
            float4 b00 = {0}, b16 = {0};
            a00 = __ldg((const float4*)(add_ptr_f(A, offsetA00)));
            a16 = __ldg((const float4*)(add_ptr_f(A, offsetA16)));
            b00 = __ldg((const float4*)(add_ptr_f(B, offsetB00)));
            b16 = __ldg((const float4*)(add_ptr_f(B, offsetB16)));

            __syncthreads();

            *(float*)&bShare[storAB + (0*32 +  0 + 0*65*32)*4] = a00.x;
            *(float*)&bShare[storAB + (1*32 +  0 + 0*65*32)*4] = a00.y;
            *(float*)&bShare[storAB + (2*32 +  0 + 0*65*32)*4] = a00.z;
            *(float*)&bShare[storAB + (3*32 +  0 + 0*65*32)*4] = a00.w;
            *(float*)&bShare[storAB + (0*32 + 16 + 0*65*32)*4] = a16.x;
            *(float*)&bShare[storAB + (1*32 + 16 + 0*65*32)*4] = a16.y;
            *(float*)&bShare[storAB + (2*32 + 16 + 0*65*32)*4] = a16.z;
            *(float*)&bShare[storAB + (3*32 + 16 + 0*65*32)*4] = a16.w;


            *(float*)&bShare[storB + (1*65*32)*4] = b00.x;
            *(float*)&bShare[storB + (1*65*32 + 1)*4] = b00.y;
            *(float*)&bShare[storB + (1*65*32 + 2)*4] = b00.z;
            *(float*)&bShare[storB + (1*65*32 + 3)*4] = b00.w;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32)*4] = b16.x;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 1)*4] = b16.y;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 2)*4] = b16.z;
            *(float*)&bShare[storB + (32*32 + 8*2 + 1*65*32 + 3)*4] = b16.w;            __syncthreads();

            float regA[8], regB[4];
            #pragma unroll
            for (int j = 0; j < 4; j++)
            {
                // fetch outer product data
                *(float4*)&regA[0] = *(float4*)&bShare[loadA + (32*j +  0)*4];
                *(float4*)&regA[4] = *(float4*)&bShare[loadA + (32*j + 16)*4];
                *(float4*)&regB[0] = *(float4*)&bShare[loadB + (32*j + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
            #pragma unroll
            for (int j = 4; j < 8; j++)
            {
                *(float2*)&regA[0] = *(float2*)&bShare[loadA + (32*j +  0 + (j/4)*2)*4];
                *(float2*)&regA[2] = *(float2*)&bShare[loadA + (32*j +  2 + (j/4)*2)*4];
                *(float2*)&regA[4] = *(float2*)&bShare[loadA + (32*j + 16 + (j/4)*2)*4];
                *(float2*)&regA[6] = *(float2*)&bShare[loadA + (32*j + 18 + (j/4)*2)*4];
                *(float2*)&regB[0] = *(float2*)&bShare[loadB + (32*j +  0 + (j/4)*2 + 65*32)*4];
                *(float2*)&regB[2] = *(float2*)&bShare[loadB + (32*j +  2 + (j/4)*2 + 65*32)*4];

                for (int i = 0; i < 8; i++)
                    for (int j = 0; j < 4; j++)
                        regC[i][j] += regA[i] * regB[j];
            }
        }

        asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid)   :);
        asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(bx)   :);
        asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(by) :);

        ty = ((tid & 16) >> 3) + (tid & 1);
        tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

        uint storC = ty*32*8*4 + tx*4;

        tx = tid % 16;
        ty = tid / 16;

        uint readC = ty*32*8 + tx*2 + ((tid & 192)>>2);

        // uint blk_index = block_index[blockIdx.x] / 2;
        // uint intra_blk_index = block_index[blockIdx.x] % 2;
        // C_val += 32 * 64 * blk_index + intra_blk_index * 32;
        // C_val += ty * 64 + tx * 2;
        // TODO double check here!
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("output offset: %d\n", (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2);
        // }

        C += (blockIdx.y * BLOCK_SIZE_M + ty) * N + blockIdx.x * BLOCK_SIZE_N + tx *2;
        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[0];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[1];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[2];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[3];
        __syncthreads();

        float2 c2[8];
        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        //-> store((bhalf2*)C, c2[0]);
        // *(float2*)C_val = c2[0];
        *(float2*)C = c2[0];
        // if(threadIdx.x==0 && blockIdx.z==1 && by==0 && bx==0){
        //     printf("output value: %f\n", *output);
        // }

        __syncthreads();
        *(float4*)&fShare[storC + 0*32*8] = *(float4*)regC[4];
        *(float4*)&fShare[storC + 1*32*8] = *(float4*)regC[5];
        *(float4*)&fShare[storC + 2*32*8] = *(float4*)regC[6];
        *(float4*)&fShare[storC + 3*32*8] = *(float4*)regC[7];
        __syncthreads();

        for (int i = 0; i < 8; i++)
            c2[i] = *(float2*)&fShare[readC + i*32];

        // Tree reduce
        for (int j = 4; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                c2[i] = _add(c2[i], c2[i+j]);

        C += 16 * N;
        // *(float2*)C_val = c2[0];
        *(float2*)C = c2[0];

    // }
    
}

void elastic_backward_function(float * activation,
                                float * weight,
                                float * grad_c,
                                float * a_grad,
                                float * w_grad,
                                int M,
                                int K,
                                int N,
                                int ori_in_features,
                                int ori_out_features)
{

    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 64;
    const int BLOCK_SIZE_N = 32;
    /*
    // ori_out_features is on the M dim
    // ori_in_features is on the N dim
    grad_w: full shape: out_features x in_features
        effective shape: N x K
    grad_w = grad_C ^T * A
             N * M x M * K

    */
    dim3 w_block_dim(256);
    dim3 w_grid_dim(ori_in_features/BLOCK_SIZE_N, ori_out_features/BLOCK_SIZE_M);
    grad_w_kernel<<<w_grid_dim, w_block_dim>>>(grad_c, activation, w_grad, N, M, K, ori_in_features, ori_out_features);
    /*
    grad_a = grad_c * weight 
            (M * N) * (N * K))
    */
   dim3 a_block_dim(256);
   dim3 a_grid_dim(K/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
   grad_a_kernel<<<a_grid_dim, a_block_dim>>>(grad_c, weight, a_grad, M, N, K, ori_in_features, ori_out_features);

}

vector<at::Tensor> elastic_sparse_linear_backward(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor grad_c,
    int in_features,
    int out_features)
{
    /*
    Compute the gradient of the Q, K, V.
    A * B = C
        |  backward()
        V
    Grad_A = Grad_C * B^T
    Grad_B = A^T * Grad_C
    */
    hipSetDevice(activation.get_device());
    torch::Tensor a_grad = torch::empty_like(activation);
    torch::Tensor w_grad = torch::zeros_like(weight);
    int batch_size = activation.size(0);
    int max_seq_len = activation.size(1);
    int ori_in_features = weight.size(1);
    int ori_out_features = weight.size(0);
    int M = max_seq_len * batch_size;
    int K = in_features;
    int N = out_features;
    assert(in_features % 64==0);
    assert(out_features % 32==0);
    AT_DISPATCH_FLOATING_TYPES(activation.type(), "seqlen_dynamic_sparse_linear", ([&]
                                {       elastic_backward_function(
                                        activation.data_ptr<float>(),
                                        weight.data_ptr<float>(),
                                        grad_c.data_ptr<float>(),
                                        a_grad.data_ptr<float>(),
                                        w_grad.data_ptr<float>(),
                                        M, K, N,
                                        ori_in_features,
                                        ori_out_features
                                    ); }));
    

    vector<torch::Tensor> grads({a_grad, w_grad});
    return grads;
}