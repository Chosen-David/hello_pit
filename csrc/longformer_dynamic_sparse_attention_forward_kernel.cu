#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
using namespace std;
// Macro definition for the cuda and cusparse

#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#define OFFSET(row, col, ld) ((row)*ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int *>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4 *>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4 *>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int *>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024
#define FULL_MASK 0xffffffff

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int *sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__device__ __forceinline__ const int *add_ptr_u(const int *src, int offset)
{
    const int *dst;
    asm("{                       \n\t"
        ".reg .u32 lo,hi,of;     \n\t"
        "mul.lo.u32 of, %2, %3;  \n\t"
        "mov.b64    {lo,hi}, %1; \n\t"
        "add.cc.u32  lo,lo,  of; \n\t"
        "addc.u32    hi,hi,  0;  \n\t"
        "mov.b64 %0, {lo,hi};    \n\t"
        "}"
        : "=l"(dst)
        : "l"(src), "r"(offset), "r"((int)sizeof(*src)));
    return dst;
}

__device__ __forceinline__ const float *add_ptr_f(const float *src, int offset)
{
    const float *dst;
    asm("{                       \n\t"
        ".reg .u32 lo,hi,of;     \n\t"
        "mul.lo.u32 of, %2, %3;  \n\t"
        "mov.b64    {lo,hi}, %1; \n\t"
        "add.cc.u32  lo,lo,  of; \n\t"
        "addc.u32    hi,hi,  0;  \n\t"
        "mov.b64 %0, {lo,hi};    \n\t"
        "}"
        : "=l"(dst)
        : "l"(src), "r"(offset), "r"((int)sizeof(*src)));
    return dst;
}

__device__ __forceinline__ float2 _add(float2 x, float2 y)
{
    float2 res;
    res.x = x.x + y.x;
    res.y = x.y + y.y;
    return res;
}

__global__ void BLOCK_SPARSE_MATMUL_OUT_32_64_32(
    float *A,
    float *B,
    float *C_val,
    int *row_index,
    int *col_index,
    int GLOBAL_M,
    int GLOBAL_K,
    int GLOBAL_N,
    int SPARSE_VAL_SIZE)
{
    /*
    description:
    tiling k dimension
    smm_dd_s_nn: sparse matmul, dense (MxK, along K) x dense (KxN, along N) -> sparse (MxN, along N)
    the output sparse is block size 32x32, the blocks will be written to bcsr 32x64
    */
    
    const int BLOCK_SIZE_M = 32; // 64
    const int BLOCK_SIZE_K = 64; // 8
    const int BLOCK_SIZE_N = 32; // 128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;

    A += M * K * blockIdx.y;
    B += K * N * blockIdx.y;
    C_val += SPARSE_VAL_SIZE * blockIdx.y;
    // if(threadIdx.x==0 && blockIdx.x==0){
    //     printf("blockIdx.x:%d blockIdx.y:%d\n", blockIdx.x, blockIdx.y);
    // }
    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    char *bShare = (char *)fShare;

    uint tid = threadIdx.x;
    uint bx = col_index[blockIdx.x]; // N
    uint by = row_index[blockIdx.x]; // M

    uint tx = tid % 16;
    uint ty = tid / 16;
    assert(THREAD_SIZE_K % 16 == 0);
    uint k = tx * 4;
    uint ori_offsetA00 = (by * 32 + ty) * K + k;
    uint ori_offsetA16 = ori_offsetA00 + K * 16;
    uint ori_offsetB00 = (bx * 32 + ty) * K + k;
    uint ori_offsetB16 = ori_offsetB00 + K * 16;

    uint tid224 = tid & 224;
    uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
    uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
    uint loadB = ((tid >> 1) & 7) << 4;
    loadA += (tid224 * 32) + (tid224 / 2);
    loadB += (tid224 * 32) + (tid224 / 2);

    // This keeps all prior logic outside of the loops.
    asm("mov.b32 %0, %0;"
        : "+r"(storAB)
        :);
    asm("mov.b32 %0, %0;"
        : "+r"(loadA)
        :);
    asm("mov.b32 %0, %0;"
        : "+r"(loadB)
        :);

    float regC[8][4];
    for (int i = 0; i < 8; i++)
        for (int j = 0; j < 4; j++)
            regC[i][j] = 0.0f;

    for (int k_seq = 0; k_seq < (int)(K / 64); k_seq++)
    {
        uint offsetA00 = ori_offsetA00 + 64 * k_seq;
        uint offsetA16 = ori_offsetA16 + 64 * k_seq;
        uint offsetB00 = ori_offsetB00 + 64 * k_seq;
        uint offsetB16 = ori_offsetB16 + 64 * k_seq;

        float4 a00 = {0}, a16 = {0};
        float4 b00 = {0}, b16 = {0};
        a00 = __ldg((const float4 *)(add_ptr_f(A, offsetA00)));
        a16 = __ldg((const float4 *)(add_ptr_f(A, offsetA16)));
        b00 = __ldg((const float4 *)(add_ptr_f(B, offsetB00)));
        b16 = __ldg((const float4 *)(add_ptr_f(B, offsetB16)));

        __syncthreads();

        *(float *)&bShare[storAB + (0 * 32 + 0 + 0 * 65 * 32) * 4] = a00.x;
        *(float *)&bShare[storAB + (1 * 32 + 0 + 0 * 65 * 32) * 4] = a00.y;
        *(float *)&bShare[storAB + (2 * 32 + 0 + 0 * 65 * 32) * 4] = a00.z;
        *(float *)&bShare[storAB + (3 * 32 + 0 + 0 * 65 * 32) * 4] = a00.w;
        *(float *)&bShare[storAB + (0 * 32 + 16 + 0 * 65 * 32) * 4] = a16.x;
        *(float *)&bShare[storAB + (1 * 32 + 16 + 0 * 65 * 32) * 4] = a16.y;
        *(float *)&bShare[storAB + (2 * 32 + 16 + 0 * 65 * 32) * 4] = a16.z;
        *(float *)&bShare[storAB + (3 * 32 + 16 + 0 * 65 * 32) * 4] = a16.w;

        *(float *)&bShare[storAB + (0 * 32 + 0 + 1 * 65 * 32) * 4] = b00.x;
        *(float *)&bShare[storAB + (1 * 32 + 0 + 1 * 65 * 32) * 4] = b00.y;
        *(float *)&bShare[storAB + (2 * 32 + 0 + 1 * 65 * 32) * 4] = b00.z;
        *(float *)&bShare[storAB + (3 * 32 + 0 + 1 * 65 * 32) * 4] = b00.w;
        *(float *)&bShare[storAB + (0 * 32 + 16 + 1 * 65 * 32) * 4] = b16.x;
        *(float *)&bShare[storAB + (1 * 32 + 16 + 1 * 65 * 32) * 4] = b16.y;
        *(float *)&bShare[storAB + (2 * 32 + 16 + 1 * 65 * 32) * 4] = b16.z;
        *(float *)&bShare[storAB + (3 * 32 + 16 + 1 * 65 * 32) * 4] = b16.w;
        __syncthreads();

        float regA[8], regB[4];
#pragma unroll
        for (int j = 0; j < 4; j++)
        {
            // fetch outer product data
            *(float4 *)&regA[0] = *(float4 *)&bShare[loadA + (32 * j + 0) * 4];
            *(float4 *)&regA[4] = *(float4 *)&bShare[loadA + (32 * j + 16) * 4];
            *(float4 *)&regB[0] = *(float4 *)&bShare[loadB + (32 * j + 65 * 32) * 4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
#pragma unroll
        for (int j = 4; j < 8; j++)
        {
            *(float2 *)&regA[0] = *(float2 *)&bShare[loadA + (32 * j + 0 + (j / 4) * 2) * 4];
            *(float2 *)&regA[2] = *(float2 *)&bShare[loadA + (32 * j + 2 + (j / 4) * 2) * 4];
            *(float2 *)&regA[4] = *(float2 *)&bShare[loadA + (32 * j + 16 + (j / 4) * 2) * 4];
            *(float2 *)&regA[6] = *(float2 *)&bShare[loadA + (32 * j + 18 + (j / 4) * 2) * 4];
            *(float2 *)&regB[0] = *(float2 *)&bShare[loadB + (32 * j + 0 + (j / 4) * 2 + 65 * 32) * 4];
            *(float2 *)&regB[2] = *(float2 *)&bShare[loadB + (32 * j + 2 + (j / 4) * 2 + 65 * 32) * 4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
    }

    asm volatile("mov.u32 %0, %tid.x;"
                 : "=r"(tid)
                 :);
    asm volatile("mov.u32 %0, %ctaid.x;"
                 : "=r"(bx)
                 :);
    asm volatile("mov.u32 %0, %ctaid.y;"
                 : "=r"(by)
                 :);

    ty = ((tid & 16) >> 3) + (tid & 1);
    tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

    uint storC = ty * 32 * 8 * 4 + tx * 4;

    tx = tid % 16;
    ty = tid / 16;

    uint readC = ty * 32 * 8 + tx * 2 + ((tid & 192) >> 2);

    // uint blk_index = block_index[blockIdx.x] / 2;
    uint blk_index = blockIdx.x;
    // uint intra_blk_index = block_index[blockIdx.x] % 2;
    // C_val += 32 * 32 * blk_index;
    // C_val += ty * 32 + tx * 2;
    // C_val += ((blockIdx.x / (GLOBAL_N / BLOCK_SIZE_N)) * BLOCK_SIZE_M + ty) * GLOBAL_N + (blockIdx.x % (GLOBAL_N / BLOCK_SIZE_N)) * BLOCK_SIZE_N + tx * 2;
    // if(threadIdx.x==0 && blockIdx.x==0){
    //     printf("blockIdx.x:%d blockIdx.y:%d\n", blockIdx.x, blockIdx.y);
    // }
    C_val += (row_index[blockIdx.x]  * BLOCK_SIZE_M + ty) * GLOBAL_N + col_index[blockIdx.x] * BLOCK_SIZE_N + tx * 2;


    __syncthreads();
    *(float4 *)&fShare[storC + 0 * 32 * 8] = *(float4 *)regC[0];
    *(float4 *)&fShare[storC + 1 * 32 * 8] = *(float4 *)regC[1];
    *(float4 *)&fShare[storC + 2 * 32 * 8] = *(float4 *)regC[2];
    *(float4 *)&fShare[storC + 3 * 32 * 8] = *(float4 *)regC[3];
    __syncthreads();

    float2 c2[8];
    for (int i = 0; i < 8; i++)
        c2[i] = *(float2 *)&fShare[readC + i * 32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i + j]);

    //-> store((bhalf2*)C, c2[0]);
    *(float2 *)C_val = c2[0];

    __syncthreads();
    *(float4 *)&fShare[storC + 0 * 32 * 8] = *(float4 *)regC[4];
    *(float4 *)&fShare[storC + 1 * 32 * 8] = *(float4 *)regC[5];
    *(float4 *)&fShare[storC + 2 * 32 * 8] = *(float4 *)regC[6];
    *(float4 *)&fShare[storC + 3 * 32 * 8] = *(float4 *)regC[7];
    __syncthreads();

    for (int i = 0; i < 8; i++)
        c2[i] = *(float2 *)&fShare[readC + i * 32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i + j]);

    // C_val += 16 * 32;
    C_val += 16 * GLOBAL_N;

    *(float2 *)C_val = c2[0];
}

void batch_matmul_block_sparse_out_kernel_launch(
    float *A,
    float *W,
    int *row_pos,
    int *col,
    float *output,
    int M, int K, int N,
    int block_h,
    int block_w,
    int block_nnz,
    int batch_size,
    int head_num)
{
    const dim3 dimBlock(256);
    const dim3 dimGrid(block_nnz, head_num * batch_size);
    BLOCK_SPARSE_MATMUL_OUT_32_64_32<<<dimGrid, dimBlock>>>(A, W, output, row_pos, col, M, K, N,  M * N);
}

at::Tensor batch_matmul_block_sparse_out(
    torch::Tensor A,
    torch::Tensor W,
    torch::Tensor row_pos,
    torch::Tensor col,
    torch::Tensor output,
    int block_h, int block_w, int block_nnz)
{
    hipSetDevice(A.get_device());
    int batch_size = A.size(0);
    int head_num = A.size(1);
    int max_seq_length = A.size(2);
    int hidden_dim = A.size(3);
    // printf("block nnz: %d \n", block_nnz);
    AT_DISPATCH_FLOATING_TYPES(A.type(), "longformer_batch_matmul", ([&]
            { batch_matmul_block_sparse_out_kernel_launch(
                    A.data_ptr<float>(),
                    W.data_ptr<float>(),
                    row_pos.data_ptr<int>(),
                    col.data_ptr<int>(),
                    output.data_ptr<float>(),
                    max_seq_length,
                    hidden_dim,
                    max_seq_length,
                    block_h,
                    block_w,
                    block_nnz,
                    batch_size,
                    head_num); }));
    return output;
}

__global__ void longformer_mixed_softmax_kernel(float * A,
                                     int * row,
                                     int *col,
                                     float* val_mask,
                                     int * global_attention,
                                     float* extra_buffer,
                                     int block_h,
                                     int block_w,
                                     int block_nnz,
                                     int row_tile,
                                     int M,
                                     int N,
                                     int global_attention_size)
{
    /*
    description:
    each row of blocks is dealt with a thread group
    each block is 32x32
    */
    A += M * N * blockIdx.y;
    extra_buffer += M * global_attention_size * blockIdx.y;
    uint blk_row_idx = blockIdx.x / (block_h/row_tile) ;
    int block_inter_row = (blockIdx.x % (block_h/row_tile)) * row_tile;
    uint bm = threadIdx.x / block_w;
    uint bn = threadIdx.x % block_w;
    assert(block_w % 32==0);
    float regC = 0.0f;
    float regSum = 0.0f;
    float regMax = -100000.0;
    int block_seq_start = row[blk_row_idx];
    int block_seq_end = row[blk_row_idx+1];
    uint A_index, col_idx, mask_index;
    for(int i=bn; i<global_attention_size; i+=32){
        A_index = (blockIdx.x * row_tile + bm) * N + global_attention[i];
        regMax = max(regMax, A[A_index]);
    }
    for (int block_seq = block_seq_start; block_seq < block_seq_end; block_seq++) {
        mask_index = block_h * block_w * block_seq + (block_inter_row + bm) * block_w + bn;
        A_index = (blockIdx.x * row_tile + bm) * N + (col[block_seq] * block_w + bn);
        regMax = max(regMax, A[A_index] * val_mask[mask_index]);

    }
    for (int offset = 16; offset > 0; offset /= 2) {
        regMax = max(regMax, __shfl_down_sync(FULL_MASK, regMax, offset));
    }
    regMax = __shfl_sync(FULL_MASK, regMax, 0);

    for(int i=bn; i<global_attention_size; i+=32){
        A_index = (blockIdx.x * row_tile + bm) * N + global_attention[i];
        regC = expf(A[A_index]-regMax);
        regSum += regC;
        A[A_index] = -10000.0;
        extra_buffer[(blockIdx.x * row_tile + bm)*global_attention_size+ i] = regC; 
    }
    for (int block_seq = block_seq_start; block_seq < block_seq_end; block_seq++) {
        mask_index = block_h * block_w * block_seq + (block_inter_row + bm) * block_w + bn;
        A_index = (blockIdx.x * row_tile + bm) * N + (col[block_seq] * block_w + bn);
        if (val_mask[mask_index] != 0) {
            regC = expf(A[A_index]-regMax);
            regSum += regC;
        }
    }
    for (int offset = 16; offset > 0; offset /= 2) {
        regSum += __shfl_down_sync(FULL_MASK, regSum, offset);
    }
    regSum = __shfl_sync(FULL_MASK, regSum, 0);
    // if(threadIdx.x%32==1)
    //     printf("Row %d Regsum %f  \n", block_inter_row + bm + blk_row_idx * block_h, regSum);
    for (int block_seq = block_seq_start; block_seq < block_seq_end; block_seq++) {
        mask_index = block_h * block_w * block_seq + (block_inter_row + bm) * block_w + bn;
        A_index = (blockIdx.x * row_tile + bm) * N + (col[block_seq] * block_w + bn);
        regC = 0.0f;
        if (val_mask[mask_index] > 0) {
            A[A_index] = expf(A[A_index]-regMax)/regSum;
        }
        else{
            A[A_index] = 0;
        }

    }
    for(int i=bn; i<global_attention_size; i+=32){
        A_index = (blockIdx.x * row_tile + bm) * N + global_attention[i];
        A[A_index] = extra_buffer[(blockIdx.x * row_tile + bm)*global_attention_size+ i]/regSum; 
    }


}
void longformer_mixed_softmax_launch(float * A,
                                     int * row,
                                     int *col,
                                     float* val_mask,
                                     int * global_attention,
                                     float* extra_buffer,
                                     int block_h,
                                     int block_w,
                                     int block_nnz,
                                     int M,
                                     int N,
                                     int head_num,
                                     int batch_size,
                                     int global_attention_size
)
{
    const int row_tile=8;
    const dim3 blockDim(row_tile*32);
    const dim3 gridDim(M/row_tile, head_num*batch_size);
    longformer_mixed_softmax_kernel<<<gridDim, blockDim>>>(A,
                                                           row,
                                                           col,
                                                           val_mask,
                                                           global_attention,
                                                           extra_buffer,
                                                           block_h,
                                                           block_w,
                                                           block_nnz,
                                                           row_tile,
                                                           M, N,
                                                           global_attention_size
                                                           );
}
at::Tensor longformer_mixed_softmax(
    torch::Tensor A,
    torch::Tensor row,
    torch::Tensor col,
    torch::Tensor val_mask,
    torch::Tensor global_attention,
    torch::Tensor extra_buffer,
    int block_h, int block_w, int block_nnz

)
{
    hipSetDevice(A.get_device());
    int batch_size = A.size(0);
    int head_num = A.size(1);
    int M = A.size(2);
    int N = A.size(3);
    AT_DISPATCH_FLOATING_TYPES(A.type(), "longformer_mixed_softmax", ([&]
            { longformer_mixed_softmax_launch(
                    A.data_ptr<float>(),
                    row.data_ptr<int>(),
                    col.data_ptr<int>(),
                    val_mask.data_ptr<float>(),
                    global_attention.data_ptr<int>(),
                    extra_buffer.data_ptr<float>(),
                    block_h,
                    block_w,
                    block_nnz,
                    M,
                    N,
                    head_num,
                    batch_size,
                    global_attention.size(0)
                    ); }));
    return A;
}


template <
    const int BLOCK_SIZE_M, // 64
    const int BLOCK_SIZE_K, // 8
    const int BLOCK_SIZE_N, // 128
    const int THREAD_SIZE_M, // 8
    const int THREAD_SIZE_K, // 4
    const int THREAD_SIZE_N  // 8
>
__global__ void BLOCK_SPARSE_MATMUL_DSD(int* csr_row, int * csr_col, float* csr_val, float * B, float* C,  int M, int K, int N, int sparse_val_size){
    // const int BLOCK_SIZE_M = 32;
    // const int BLOCK_SIZE_K = 32;
    // const int BLOCK_SIZE_N = 64;
    // const int THREAD_SIZE_M = 4;
    // const int THREAD_SIZE_K = 4;
    // const int THREAD_SIZE_N = 4;
    int by = blockIdx.y; // M
    int bx = blockIdx.x; // N
    int bz = blockIdx.z;
    int ty = threadIdx.y; 
    int tx = threadIdx.x;
    csr_val = csr_val + sparse_val_size * bz;
    B = B + K * N * bz;
    C = C + M * N * bz;

    const int padding = 1;
    __shared__ float As[BLOCK_SIZE_M * (padding+BLOCK_SIZE_K)];
    __shared__ float Bs[BLOCK_SIZE_N * (padding+BLOCK_SIZE_K)];

    float accum[THREAD_SIZE_N][THREAD_SIZE_M] = {0};
    float a_frag[THREAD_SIZE_M][THREAD_SIZE_K];
    float b_frag[THREAD_SIZE_N][THREAD_SIZE_K];

    int A_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    int B_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    int bszy = BLOCK_SIZE_M / THREAD_SIZE_M;
    int bszx = BLOCK_SIZE_N / THREAD_SIZE_N;

    int THREADS_PER_BLOCK = bszy * bszx;

    int A_TILE_ROW_STRIDE = THREADS_PER_BLOCK / A_THREAD_PER_ROW;
    int B_TILE_ROW_STRIDE = THREADS_PER_BLOCK / B_THREAD_PER_ROW;

    int tid = ty * bszx + tx;

    int index_start = csr_row[by], index_end = csr_row[by+1];

    int A_BLOCK_ROW_START = tid / A_THREAD_PER_ROW;
    int B_BLOCK_ROW_START = tid / B_THREAD_PER_ROW;

    int A_BLOCK_COL_START = tid % A_THREAD_PER_ROW * 4;
    int B_BLOCK_COL_START = tid % B_THREAD_PER_ROW * 4;
    const int vBLOCK_SIZE_M = BLOCK_SIZE_M / THREAD_SIZE_M;
    const int vBLOCK_SIZE_N = BLOCK_SIZE_N / THREAD_SIZE_N;

    for(int tile_block_idx = index_start; tile_block_idx < index_end; tile_block_idx += 1){
        int col_pos = csr_col[tile_block_idx] * BLOCK_SIZE_K;
        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_M; k += A_TILE_ROW_STRIDE){
            FETCH_FLOAT4(As[OFFSET(k+A_BLOCK_ROW_START, A_BLOCK_COL_START, BLOCK_SIZE_K)]) =
                FETCH_FLOAT4(csr_val[OFFSET(k + A_BLOCK_ROW_START + by*BLOCK_SIZE_M, A_BLOCK_COL_START + col_pos, K)]);
                // FETCH_FLOAT4(csr_val[tile_block_idx * BLOCK_SIZE_M * BLOCK_SIZE_K + OFFSET(k+A_BLOCK_ROW_START, A_BLOCK_COL_START, BLOCK_SIZE_K)]);
        }

        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_K; k += B_TILE_ROW_STRIDE){
            FETCH_FLOAT4(Bs[OFFSET(k+B_BLOCK_ROW_START, B_BLOCK_COL_START, BLOCK_SIZE_N)]) = 
                FETCH_FLOAT4(B[OFFSET(col_pos+k+B_BLOCK_ROW_START, bx*BLOCK_SIZE_N + B_BLOCK_COL_START, N)]);
                // FETCH_FLOAT4(W_val[tile_block_idx * BLOCK_SIZE_N * BLOCK_SIZE_K + (k+B_BLOCK_ROW_START) * BLOCK_SIZE_N + B_BLOCK_COL_START]);
                // FETCH_FLOAT4(B[OFFSET(tile_idx+k+B_BLOCK_ROW_START, bx*BLOCK_SIZE_N+B_BLOCK_COL_START, N)]);
        }

        __syncthreads();

        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_K; k += THREAD_SIZE_K){
            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_K; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_M; j += 1){
                    a_frag[j][i] = As[OFFSET(ty + vBLOCK_SIZE_M * j, k+i, BLOCK_SIZE_K)];
                    //a_frag[j][i] = As[OFFSET(k+i, ty + vBLOCK_SIZE_M * j, BLOCK_SIZE_M)];
                }
            }

            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_K; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_N; j += 1){
                    b_frag[j][i] = Bs[OFFSET(k+i, tx + vBLOCK_SIZE_N * j, BLOCK_SIZE_N)];
                }
            }

            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_N; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_M; j++){
                    #pragma unroll
                    for(int k_in = 0; k_in < THREAD_SIZE_K; k_in++){
                        // accum[i][j] = fma(a_frag[j][k_in], b_frag[i][k_in], accum[i][j]);
                        accum[i][j] += a_frag[j][k_in] * b_frag[i][k_in];
                    }
                }
            }
        }

        __syncthreads();
    }


    #pragma unroll
    for(int thread_x = 0; thread_x < THREAD_SIZE_N; thread_x++){
        #pragma unroll
        for(int thread_y = 0; thread_y < THREAD_SIZE_M; thread_y+=1){
            C[OFFSET(
                BLOCK_SIZE_M * by + ty + thread_y * vBLOCK_SIZE_M,
                BLOCK_SIZE_N * bx + tx + thread_x * vBLOCK_SIZE_N,
                N
            )] = (accum[thread_x][thread_y]);
        }
    }


}


void batch_matmul_block_sparse_kernel_launch(
    float * A,
    float * B,
    float * C,
    int * row_ptr,
    int * col_idx,
    int M,
    int K,
    int N,
    int head_num,
    int batch_size,
    int block_h,
    int block_w

)
{
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 64;
    const int THREAD_SIZE_M = 4;
    const int THREAD_SIZE_K = 4;
    const int THREAD_SIZE_N = 4;
    assert(block_h==BLOCK_SIZE_M);
    assert(block_w==BLOCK_SIZE_K);
    dim3 gridDim(N/BLOCK_SIZE_N, M/BLOCK_SIZE_M, head_num * batch_size);
    dim3 blockDim(BLOCK_SIZE_N/THREAD_SIZE_N, BLOCK_SIZE_M/THREAD_SIZE_M);
    BLOCK_SPARSE_MATMUL_DSD<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_M, THREAD_SIZE_K, THREAD_SIZE_N><<<gridDim, blockDim>>>(
        row_ptr,
        col_idx,
        A,
        B,
        C,
        M,
        K,
        N,
        M*K);

}

at::Tensor batch_matmul_block_sparse(
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor row_ptr,
    torch::Tensor col_idx,
    int block_h,
    int block_w
){
    hipSetDevice(A.get_device());
    int batch_size = A.size(0);
    int head_num = A.size(1);
    int M = A.size(2);
    int K = A.size(3);
    int N = B.size(3);
    // printf("M:%d N:%d\n", M, N);
    torch::Tensor output = torch::zeros({batch_size, head_num, M, N}, A.options());
    AT_DISPATCH_FLOATING_TYPES(A.type(), "longformer_batch_matmul_attenxV", ([&]
            { batch_matmul_block_sparse_kernel_launch(
                A.data_ptr<float>(),
                B.data_ptr<float>(),
                output.data_ptr<float>(),
                row_ptr.data_ptr<int>(),
                col_idx.data_ptr<int>(),
                M,
                K,
                N,
                head_num,
                batch_size,
                block_h,
                block_w); }));
    return output;
}
