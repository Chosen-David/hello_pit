#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
using namespace std;
// Macro definition for the cuda and cusparse

#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#define OFFSET(row, col, ld) ((row)*ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int *>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4 *>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4 *>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int *>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024
#define FULL_MASK 0xffffffff

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int *sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__device__ __forceinline__ const int *add_ptr_u(const int *src, int offset)
{
    const int *dst;
    asm("{                       \n\t"
        ".reg .u32 lo,hi,of;     \n\t"
        "mul.lo.u32 of, %2, %3;  \n\t"
        "mov.b64    {lo,hi}, %1; \n\t"
        "add.cc.u32  lo,lo,  of; \n\t"
        "addc.u32    hi,hi,  0;  \n\t"
        "mov.b64 %0, {lo,hi};    \n\t"
        "}"
        : "=l"(dst)
        : "l"(src), "r"(offset), "r"((int)sizeof(*src)));
    return dst;
}

__device__ __forceinline__ const float *add_ptr_f(const float *src, int offset)
{
    const float *dst;
    asm("{                       \n\t"
        ".reg .u32 lo,hi,of;     \n\t"
        "mul.lo.u32 of, %2, %3;  \n\t"
        "mov.b64    {lo,hi}, %1; \n\t"
        "add.cc.u32  lo,lo,  of; \n\t"
        "addc.u32    hi,hi,  0;  \n\t"
        "mov.b64 %0, {lo,hi};    \n\t"
        "}"
        : "=l"(dst)
        : "l"(src), "r"(offset), "r"((int)sizeof(*src)));
    return dst;
}

__device__ __forceinline__ float2 _add(float2 x, float2 y)
{
    float2 res;
    res.x = x.x + y.x;
    res.y = x.y + y.y;
    return res;
}

__global__ void BLOCK_SPARSE_MATMUL_OUT_32_64_32(
    float *A,
    float *B,
    float *C_val,
    int *row_index,
    int *col_index,
    int GLOBAL_M,
    int GLOBAL_K,
    int GLOBAL_N,
    int SPARSE_VAL_SIZE)
{
    /*
    description:
    tiling k dimension
    smm_dd_s_nn: sparse matmul, dense (MxK, along K) x dense (KxN, along N) -> sparse (MxN, along N)
    the output sparse is block size 32x32, the blocks will be written to bcsr 32x64
    */
    
    const int BLOCK_SIZE_M = 32; // 64
    const int BLOCK_SIZE_K = 64; // 8
    const int BLOCK_SIZE_N = 32; // 128
    const int THREAD_SIZE_K = 64;
    const int M = GLOBAL_M;
    const int K = GLOBAL_K;
    const int N = GLOBAL_N;

    A += M * K * blockIdx.y;
    B += K * N * blockIdx.y;
    C_val += SPARSE_VAL_SIZE * blockIdx.y;
    // if(threadIdx.x==0 && blockIdx.x==0){
    //     printf("blockIdx.x:%d blockIdx.y:%d\n", blockIdx.x, blockIdx.y);
    // }
    assert(blockDim.x % 32 == 0);
    uint n_warp = 8; // blockDim.x / 32
    assert(THREAD_SIZE_K % n_warp == 0);
    // THREAD_SIZE_K: one loop k
    assert(K % THREAD_SIZE_K == 0);

    assert(BLOCK_SIZE_M == BLOCK_SIZE_N);
    __shared__ float fShare[65 * 32 * 2];
    char *bShare = (char *)fShare;

    uint tid = threadIdx.x;
    uint bx = col_index[blockIdx.x]; // N
    uint by = row_index[blockIdx.x]; // M

    uint tx = tid % 16;
    uint ty = tid / 16;
    assert(THREAD_SIZE_K % 16 == 0);
    uint k = tx * 4;
    uint ori_offsetA00 = (by * 32 + ty) * K + k;
    uint ori_offsetA16 = ori_offsetA00 + K * 16;
    uint ori_offsetB00 = (bx * 32 + ty) * K + k;
    uint ori_offsetB16 = ori_offsetB00 + K * 16;

    uint tid224 = tid & 224;
    uint storAB = (tx * 32 * 4 + ty + tx * 2) * 4;
    uint loadA = (((tid & 16) >> 3) | (tid & 1)) << 4;
    uint loadB = ((tid >> 1) & 7) << 4;
    loadA += (tid224 * 32) + (tid224 / 2);
    loadB += (tid224 * 32) + (tid224 / 2);

    // This keeps all prior logic outside of the loops.
    asm("mov.b32 %0, %0;"
        : "+r"(storAB)
        :);
    asm("mov.b32 %0, %0;"
        : "+r"(loadA)
        :);
    asm("mov.b32 %0, %0;"
        : "+r"(loadB)
        :);

    float regC[8][4];
    for (int i = 0; i < 8; i++)
        for (int j = 0; j < 4; j++)
            regC[i][j] = 0.0f;

    for (int k_seq = 0; k_seq < (int)(K / 64); k_seq++)
    {
        uint offsetA00 = ori_offsetA00 + 64 * k_seq;
        uint offsetA16 = ori_offsetA16 + 64 * k_seq;
        uint offsetB00 = ori_offsetB00 + 64 * k_seq;
        uint offsetB16 = ori_offsetB16 + 64 * k_seq;

        float4 a00 = {0}, a16 = {0};
        float4 b00 = {0}, b16 = {0};
        a00 = __ldg((const float4 *)(add_ptr_f(A, offsetA00)));
        a16 = __ldg((const float4 *)(add_ptr_f(A, offsetA16)));
        b00 = __ldg((const float4 *)(add_ptr_f(B, offsetB00)));
        b16 = __ldg((const float4 *)(add_ptr_f(B, offsetB16)));

        __syncthreads();

        *(float *)&bShare[storAB + (0 * 32 + 0 + 0 * 65 * 32) * 4] = a00.x;
        *(float *)&bShare[storAB + (1 * 32 + 0 + 0 * 65 * 32) * 4] = a00.y;
        *(float *)&bShare[storAB + (2 * 32 + 0 + 0 * 65 * 32) * 4] = a00.z;
        *(float *)&bShare[storAB + (3 * 32 + 0 + 0 * 65 * 32) * 4] = a00.w;
        *(float *)&bShare[storAB + (0 * 32 + 16 + 0 * 65 * 32) * 4] = a16.x;
        *(float *)&bShare[storAB + (1 * 32 + 16 + 0 * 65 * 32) * 4] = a16.y;
        *(float *)&bShare[storAB + (2 * 32 + 16 + 0 * 65 * 32) * 4] = a16.z;
        *(float *)&bShare[storAB + (3 * 32 + 16 + 0 * 65 * 32) * 4] = a16.w;

        *(float *)&bShare[storAB + (0 * 32 + 0 + 1 * 65 * 32) * 4] = b00.x;
        *(float *)&bShare[storAB + (1 * 32 + 0 + 1 * 65 * 32) * 4] = b00.y;
        *(float *)&bShare[storAB + (2 * 32 + 0 + 1 * 65 * 32) * 4] = b00.z;
        *(float *)&bShare[storAB + (3 * 32 + 0 + 1 * 65 * 32) * 4] = b00.w;
        *(float *)&bShare[storAB + (0 * 32 + 16 + 1 * 65 * 32) * 4] = b16.x;
        *(float *)&bShare[storAB + (1 * 32 + 16 + 1 * 65 * 32) * 4] = b16.y;
        *(float *)&bShare[storAB + (2 * 32 + 16 + 1 * 65 * 32) * 4] = b16.z;
        *(float *)&bShare[storAB + (3 * 32 + 16 + 1 * 65 * 32) * 4] = b16.w;
        __syncthreads();

        float regA[8], regB[4];
#pragma unroll
        for (int j = 0; j < 4; j++)
        {
            // fetch outer product data
            *(float4 *)&regA[0] = *(float4 *)&bShare[loadA + (32 * j + 0) * 4];
            *(float4 *)&regA[4] = *(float4 *)&bShare[loadA + (32 * j + 16) * 4];
            *(float4 *)&regB[0] = *(float4 *)&bShare[loadB + (32 * j + 65 * 32) * 4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
#pragma unroll
        for (int j = 4; j < 8; j++)
        {
            *(float2 *)&regA[0] = *(float2 *)&bShare[loadA + (32 * j + 0 + (j / 4) * 2) * 4];
            *(float2 *)&regA[2] = *(float2 *)&bShare[loadA + (32 * j + 2 + (j / 4) * 2) * 4];
            *(float2 *)&regA[4] = *(float2 *)&bShare[loadA + (32 * j + 16 + (j / 4) * 2) * 4];
            *(float2 *)&regA[6] = *(float2 *)&bShare[loadA + (32 * j + 18 + (j / 4) * 2) * 4];
            *(float2 *)&regB[0] = *(float2 *)&bShare[loadB + (32 * j + 0 + (j / 4) * 2 + 65 * 32) * 4];
            *(float2 *)&regB[2] = *(float2 *)&bShare[loadB + (32 * j + 2 + (j / 4) * 2 + 65 * 32) * 4];

            for (int i = 0; i < 8; i++)
                for (int j = 0; j < 4; j++)
                    regC[i][j] += regA[i] * regB[j];
        }
    }

    asm volatile("mov.u32 %0, %tid.x;"
                 : "=r"(tid)
                 :);
    asm volatile("mov.u32 %0, %ctaid.x;"
                 : "=r"(bx)
                 :);
    asm volatile("mov.u32 %0, %ctaid.y;"
                 : "=r"(by)
                 :);

    ty = ((tid & 16) >> 3) + (tid & 1);
    tx = ((tid >> 1) & 7) + ((tid & 224) >> 2) + (ty << 2);

    uint storC = ty * 32 * 8 * 4 + tx * 4;

    tx = tid % 16;
    ty = tid / 16;

    uint readC = ty * 32 * 8 + tx * 2 + ((tid & 192) >> 2);

    // uint blk_index = block_index[blockIdx.x] / 2;
    uint blk_index = blockIdx.x;
    // uint intra_blk_index = block_index[blockIdx.x] % 2;
    // C_val += 32 * 32 * blk_index;
    // C_val += ty * 32 + tx * 2;
    // C_val += ((blockIdx.x / (GLOBAL_N / BLOCK_SIZE_N)) * BLOCK_SIZE_M + ty) * GLOBAL_N + (blockIdx.x % (GLOBAL_N / BLOCK_SIZE_N)) * BLOCK_SIZE_N + tx * 2;
    // if(threadIdx.x==0 && blockIdx.x==0){
    //     printf("blockIdx.x:%d blockIdx.y:%d\n", blockIdx.x, blockIdx.y);
    // }
    C_val += (row_index[blockIdx.x]  * BLOCK_SIZE_M + ty) * GLOBAL_N + col_index[blockIdx.x] * BLOCK_SIZE_N + tx * 2;


    __syncthreads();
    *(float4 *)&fShare[storC + 0 * 32 * 8] = *(float4 *)regC[0];
    *(float4 *)&fShare[storC + 1 * 32 * 8] = *(float4 *)regC[1];
    *(float4 *)&fShare[storC + 2 * 32 * 8] = *(float4 *)regC[2];
    *(float4 *)&fShare[storC + 3 * 32 * 8] = *(float4 *)regC[3];
    __syncthreads();

    float2 c2[8];
    for (int i = 0; i < 8; i++)
        c2[i] = *(float2 *)&fShare[readC + i * 32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i + j]);

    //-> store((bhalf2*)C, c2[0]);
    *(float2 *)C_val = c2[0];

    __syncthreads();
    *(float4 *)&fShare[storC + 0 * 32 * 8] = *(float4 *)regC[4];
    *(float4 *)&fShare[storC + 1 * 32 * 8] = *(float4 *)regC[5];
    *(float4 *)&fShare[storC + 2 * 32 * 8] = *(float4 *)regC[6];
    *(float4 *)&fShare[storC + 3 * 32 * 8] = *(float4 *)regC[7];
    __syncthreads();

    for (int i = 0; i < 8; i++)
        c2[i] = *(float2 *)&fShare[readC + i * 32];

    // Tree reduce
    for (int j = 4; j > 0; j >>= 1)
        for (int i = 0; i < j; i++)
            c2[i] = _add(c2[i], c2[i + j]);

    // C_val += 16 * 32;
    C_val += 16 * GLOBAL_N;

    *(float2 *)C_val = c2[0];
}

void batch_matmul_block_sparse_kernel_launch(
    float *A,
    float *W,
    int *row_pos,
    int *col,
    float *output,
    int M, int K, int N,
    int block_h,
    int block_w,
    int block_nnz,
    int batch_size,
    int head_num)
{
    const dim3 dimBlock(256);
    const dim3 dimGrid(block_nnz, head_num * batch_size);
    BLOCK_SPARSE_MATMUL_OUT_32_64_32<<<dimGrid, dimBlock>>>(A, W, output, row_pos, col, M, K, N,  M * N);
}

at::Tensor batch_matmul_block_sparse_out(
    torch::Tensor A,
    torch::Tensor W,
    torch::Tensor row_pos,
    torch::Tensor col,
    torch::Tensor output,
    int block_h, int block_w, int block_nnz)
{
    hipSetDevice(A.get_device());
    int batch_size = A.size(0);
    int head_num = A.size(1);
    int max_seq_length = A.size(2);
    int hidden_dim = A.size(3);
    // printf("block nnz: %d \n", block_nnz);
    AT_DISPATCH_FLOATING_TYPES(A.type(), "longformer_batch_matmul", ([&]
            { batch_matmul_block_sparse_kernel_launch(
                    A.data_ptr<float>(),
                    W.data_ptr<float>(),
                    row_pos.data_ptr<int>(),
                    col.data_ptr<int>(),
                    output.data_ptr<float>(),
                    max_seq_length,
                    hidden_dim,
                    max_seq_length,
                    block_h,
                    block_w,
                    block_nnz,
                    batch_size,
                    head_num); }));
    return output;
}

__global__ void longformer_mixed_softmax_kernel(float * A,
                                     int * row,
                                     int *col,
                                     float* val_mask,
                                     int * global_attention,
                                     float* extra_buffer,
                                     int block_h,
                                     int block_w,
                                     int block_nnz,
                                     int row_tile,
                                     int M,
                                     int N,
                                     int global_attention_size)
{
    /*
    description:
    each row of blocks is dealt with a thread group
    each block is 32x32
    */
    A += M * N * blockIdx.y;
    extra_buffer += M * global_attention_size * blockIdx.y;
    uint blk_row_idx = blockIdx.x / (block_h/row_tile) ;
    int block_inter_row = (blockIdx.x % (block_h/row_tile)) * row_tile;
    uint bm = threadIdx.x / block_w;
    uint bn = threadIdx.x % block_w;
    assert(block_w % 32==0);
    float regC = 0.0f;
    float regSum = 0.0f;
    float regMax = -100000.0;
    int block_seq_start = row[blk_row_idx];
    int block_seq_end = row[blk_row_idx+1];
    uint A_index, col_idx, mask_index;
    for(int i=bn; i<global_attention_size; i+=32){
        A_index = (blockIdx.x * row_tile + bm) * N + global_attention[i];
        regMax = max(regMax, A[A_index]);
    }
    for (int block_seq = block_seq_start; block_seq < block_seq_end; block_seq++) {
        mask_index = block_h * block_w * block_seq + (block_inter_row + bm) * block_w + bn;
        A_index = (blockIdx.x * row_tile + bm) * N + (col[block_seq] * block_w + bn);
        regMax = max(regMax, A[A_index] * val_mask[mask_index]);

    }
    for (int offset = 16; offset > 0; offset /= 2) {
        regMax = max(regMax, __shfl_down_sync(FULL_MASK, regMax, offset));
    }
    regMax = __shfl_sync(FULL_MASK, regMax, 0);

    for(int i=bn; i<global_attention_size; i+=32){
        A_index = (blockIdx.x * row_tile + bm) * N + global_attention[i];
        regC = expf(A[A_index]-regMax);
        regSum += regC;
        A[A_index] = -10000.0;
        extra_buffer[(blockIdx.x * row_tile + bm)*global_attention_size+ i] = regC; 
    }
    for (int block_seq = block_seq_start; block_seq < block_seq_end; block_seq++) {
        mask_index = block_h * block_w * block_seq + (block_inter_row + bm) * block_w + bn;
        A_index = (blockIdx.x * row_tile + bm) * N + (col[block_seq] * block_w + bn);
        if (val_mask[mask_index] != 0) {
            regC = expf(A[A_index]-regMax);
            regSum += regC;
        }
    }
    for (int offset = 16; offset > 0; offset /= 2) {
        regSum += __shfl_down_sync(FULL_MASK, regSum, offset);
    }
    regSum = __shfl_sync(FULL_MASK, regSum, 0);
    // if(threadIdx.x%32==1)
    //     printf("Row %d Regsum %f  \n", block_inter_row + bm + blk_row_idx * block_h, regSum);
    for (int block_seq = block_seq_start; block_seq < block_seq_end; block_seq++) {
        mask_index = block_h * block_w * block_seq + (block_inter_row + bm) * block_w + bn;
        A_index = (blockIdx.x * row_tile + bm) * N + (col[block_seq] * block_w + bn);
        regC = 0.0f;
        if (val_mask[mask_index] > 0) {
            A[A_index] = expf(A[A_index]-regMax)/regSum;
        }
        else{
            A[A_index] = 0;
        }

    }
    for(int i=bn; i<global_attention_size; i+=32){
        A_index = (blockIdx.x * row_tile + bm) * N + global_attention[i];
        A[A_index] = extra_buffer[(blockIdx.x * row_tile + bm)*global_attention_size+ i]/regSum; 
    }


}
void longformer_mixed_softmax_launch(float * A,
                                     int * row,
                                     int *col,
                                     float* val_mask,
                                     int * global_attention,
                                     float* extra_buffer,
                                     int block_h,
                                     int block_w,
                                     int block_nnz,
                                     int M,
                                     int N,
                                     int head_num,
                                     int batch_size,
                                     int global_attention_size
)
{
    const int row_tile=8;
    const dim3 blockDim(row_tile*32);
    const dim3 gridDim(M/row_tile, head_num*batch_size);
    longformer_mixed_softmax_kernel<<<gridDim, blockDim>>>(A,
                                                           row,
                                                           col,
                                                           val_mask,
                                                           global_attention,
                                                           extra_buffer,
                                                           block_h,
                                                           block_w,
                                                           block_nnz,
                                                           row_tile,
                                                           M, N,
                                                           global_attention_size
                                                           );
}
at::Tensor longformer_mixed_softmax(
    torch::Tensor A,
    torch::Tensor row,
    torch::Tensor col,
    torch::Tensor val_mask,
    torch::Tensor global_attention,
    torch::Tensor extra_buffer,
    int block_h, int block_w, int block_nnz

)
{
    hipSetDevice(A.get_device());
    int batch_size = A.size(0);
    int head_num = A.size(1);
    int M = A.size(2);
    int N = A.size(3);
    AT_DISPATCH_FLOATING_TYPES(A.type(), "longformer_mixed_softmax", ([&]
            { longformer_mixed_softmax_launch(
                    A.data_ptr<float>(),
                    row.data_ptr<int>(),
                    col.data_ptr<int>(),
                    val_mask.data_ptr<float>(),
                    global_attention.data_ptr<int>(),
                    extra_buffer.data_ptr<float>(),
                    block_h,
                    block_w,
                    block_nnz,
                    M,
                    N,
                    head_num,
                    batch_size,
                    global_attention.size(0)
                    ); }));
    return A;
}